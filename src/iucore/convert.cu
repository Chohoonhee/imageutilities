#include "hip/hip_runtime.h"

//#include <hip/hip_vector_types.h>
#include "coredefs.h"
#include "memorydefs.h"
//#include "iutextures.cuh"
#include <assert.h>
#include <hip/hip_runtime.h>


/**
  Atomic add for double using comapre-and-swap (CAS)
  Taken from nVidia CUDA Programming Guide
  http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
  */
//__device__ double atomicAdd(double* address, double val)
//{

//    unsigned long long int* address_as_ull = (unsigned long long int*)address;
//    unsigned long long int old = *address_as_ull, assumed;
//    do
//    {
//        assumed = old;
//        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));

//    } while (assumed != old);  // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)

//    return __longlong_as_double(old);
//}

namespace iuprivate {


/* ***************************************************************************
 *  CUDA KERNELS
 * ***************************************************************************/

//-----------------------------------------------------------------------------
/** convert kernel 32f_C3 -> 32f_C4 (float3 -> float4)
 */
__global__ void cuConvertC3ToC4Kernel(const float3* src, size_t src_stride,
                                      float4* dst, size_t dst_stride, int width, int height)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int c_src = y*src_stride + x;
  int c_dst = y*dst_stride + x;

  if (x<width && y<height)
  {
    float3 val=src[c_src];
    dst[c_dst] =  make_float4(val.x, val.y, val.z, 1.0f);
  }
}

//-----------------------------------------------------------------------------
/** convert kernel 32f_C4 -> 32f_C3 (float4 -> float3)
 */
__global__ void cuConvertC4ToC3Kernel(const float4* src, size_t src_stride,
                                      float3* dst, size_t dst_stride, int width, int height)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int src_c = y*src_stride + x;
  int dst_c = y*dst_stride + x;

  if (x<width && y<height)
  {
    float4 val=src[src_c];
    dst[dst_c] = make_float3(val.x, val.y, val.z);
  }
}


//-----------------------------------------------------------------------------
/** convert kernel 8u_C1 -> 32f_C1 (unsigned char -> float)
 */
__global__ void cuConvert8uC1To32fC1Kernel(const unsigned char *src, size_t src_stride,
                                           float* dst, size_t dst_stride, float mul_constant,
                                           float add_constant, int width, int height)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int src_c = y*src_stride + x;
  int dst_c = y*dst_stride + x;

  if (x<width && y<height)
  {
    dst[dst_c] = src[src_c] * mul_constant + add_constant;
  }
}


//-----------------------------------------------------------------------------
/** convert kernel 32u_C1 -> 32f_C1 (unsigned int -> float)
 */
__global__ void cuConvert32uC1To32fC1Kernel(const unsigned int *src, size_t src_stride,
                                           float* dst, size_t dst_stride, float mul_constant,
                                           float add_constant, int width, int height)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int src_c = y*src_stride + x;
  int dst_c = y*dst_stride + x;

  if (x<width && y<height)
  {
    dst[dst_c] = src[src_c] * mul_constant + add_constant;
  }
}



/** convert kernel 8u_C3 -> 32f_C4 (unsigned char -> float)
 */
__global__ void cuConvert8uC3To32fC4Kernel(const unsigned char *src, size_t src_pitch,
                                           float4* dst, size_t dst_stride, float mul_constant,
                                           float add_constant, int width, int height)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int src_c = y*src_pitch + x*3;
  int dst_c = y*dst_stride + x;

  if (x<width && y<height)
  {
    dst[dst_c] = make_float4(src[src_c]/255.0f, src[src_c+1]/255.0f, src[src_c+2]/255.0f, 1.0f);// * mul_constant + add_constant;
  }
}


//-----------------------------------------------------------------------------
/** convert kernel 32f_C1 -> 8u_C1 (float -> unsigned char)
 */
__global__ void cuConvert32fC1To8uC1Kernel(const float* src, size_t src_stride,
                                           unsigned char* dst, size_t dst_stride, float mul_constant,
                                           unsigned char add_constant, int width, int height)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int src_c = y*src_stride + x;
  int dst_c = y*dst_stride + x;

  if (x<width && y<height)
  {
    dst[dst_c] = src[src_c] * mul_constant + add_constant;
  }
}

//-----------------------------------------------------------------------------
/** convert kernel 32f_C4 -> 8u_C4 (float4 -> unsigned char4)
 */
__global__ void cuConvert32fC4To8uC4Kernel(const float4* src, size_t src_stride,
                                           uchar4* dst, size_t dst_stride, float mul_constant,
                                           unsigned char add_constant, int width, int height)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int src_c = y*src_stride + x;
  int dst_c = y*dst_stride + x;

  if (x<width && y<height)
  {
    float4 val = src[src_c];
    uchar4 res;
    res.x = val.x * mul_constant + add_constant;
    res.y = val.y * mul_constant + add_constant;
    res.z = val.z * mul_constant + add_constant;
    res.w = val.w * mul_constant + add_constant;
    dst[dst_c] = res;
  }
}

//-----------------------------------------------------------------------------
/** convert kernel rgb -> hsv
 */
__global__ void cuConvertRGBToHSVKernel(const float4* src, float4* dst, size_t stride,
                                        int width, int height, bool normalize)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int c = y*stride + x;

  if (x<width && y<height)
  {
    // Read
    float4 in = src[c];
    float R = in.x;
    float G = in.y;
    float B = in.z;

    float Ma = fmaxf(R, fmaxf(G, B));
    float mi = fminf(R, fminf(G, B));
    float C = Ma-mi;

    // Hue
    float H = 0.0f;
    if (C != 0.0f)
    {
      if (Ma == R)
        H = fmod((G - B)/C, 6.0f);
      if (Ma == G)
        H = (B - R)/C + 2.0f;
      if (Ma == B)
        H = (R - G)/C + 4.0f;
    }

    H *= 60.0f;

    // Value
    float V = Ma;

    // Saturation
    float S = 0.0f;
    if (C != 0.0f)
      S = C/V;

    if (H < 0.0f)
      H += 360.0f;

    // Normalize
    if (normalize)
      H /= 360.0f;

    // Write Back
    dst[c] = make_float4(H, S, V, in.w);
  }
}

//-----------------------------------------------------------------------------
/** convert kernel hsv -> rgb
 */
__global__ void cuConvertHSVToRGBKernel(const float4* src, float4* dst, size_t stride,
                                        int width, int height, bool denormalize)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int c = y*stride + x;

  if (x<width && y<height)
  {
    // Read
    float4 in = src[c];
    float H = in.x;
    float S = in.y;
    float V = in.z;

    float4 rgb = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

    //    float C = V*S;

    //    // Denormalize
    //    if (denormalize)
    //      H = H*360.0f;

    //    // RGB
    //    H /= 60.0f;
    //    float X = C*(1.0f - abs(fmod(H, 2.0f) - 1.0f));


    //    if (H >= 0.0f)
    //    {
    //      if (H < 1.0f)
    //        rgb = make_float4(C, X, 0.0f, 0.0f);
    //      else if (H < 2.0f)
    //        rgb = make_float4(X, C, 0.0f, 0.0f);
    //      else if (H < 3.0f)
    //        rgb = make_float4(0.0f, C, X, 0.0f);
    //      else if (H < 4.0f)
    //        rgb = make_float4(0.0f, X, C, 0.0f);
    //      else if (H < 5.0f)
    //        rgb = make_float4(X, 0.0f, C, 0.0f);
    //      else if (H <= 6.0f)
    //        rgb = make_float4(C, 0.0f, X, 0.0f);
    //    }

    //    float m = V-C;
    //    rgb += m;

    if (S == 0)
    {
      rgb = make_float4(V, V, V, in.w);
      dst[c] = rgb;
      return;
    }

     H /= 60.0f;
     int i = floor(H);
     float f = H-i;
     float p = V*(1.0f - S);
     float q = V*(1.0f - S*f);
     float t = V*(1.0f - S*(1.0f-f));

     if (i == 0)
       rgb = make_float4(V, t, p, in.w);
     else if (i == 1)
       rgb = make_float4(q, V, p, in.w);
     else if (i == 2)
       rgb = make_float4(p, V, t, in.w);
     else if (i == 3)
       rgb = make_float4(p, q, V, in.w);
     else if (i == 4)
       rgb = make_float4(t, p, V, in.w);
     else if (i == 5)
       rgb = make_float4(V, p, q, in.w);



    // Write Back
    rgb.w = in.w;
    dst[c] = rgb;
  }
}


//-----------------------------------------------------------------------------
/** convert kernel rgb -> lab
 */
__global__ void cuConvertRGBToLABKernel(const float4* src, float4* dst, size_t stride,
                                        int width, int height, bool isNormalized)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int c = y*stride + x;

  if (x<width && y<height)
  {
    // Read
    float4 in = src[c];
    if (!isNormalized)
    {
      in.x /= 255.0f;
      in.y /= 255.0f;
      in.z /= 255.0f;
      in.w /= 255.0f;
    }
    
    float R = in.x;
    float G = in.y;
    float B = in.z;

    
    // convert to XYZ
    float4 XYZ;
    XYZ.x = 0.4124564f*R + 0.3575761f*G + 0.1804375f*B;
    XYZ.y = 0.2126729f*R + 0.7151522f*G + 0.0721750f*B;
    XYZ.z = 0.0193339f*R + 0.1191920f*G + 0.9503041f*B;
    
    // normalize for D65 white point
    XYZ.x /= 0.950456f;
    XYZ.z /= 1.088754f;
    
    float cubeRootX, cubeRootY, cubeRootZ;
    const float T1 = 216/24389.0f;
    const float T2 = 24389/27.0f;
    
   if (XYZ.x > T1)
      cubeRootX = cbrtf(XYZ.x);
    else
      cubeRootX = (T2 * XYZ.x + 16) / 116;
    
    if (XYZ.y > T1)
      cubeRootY = cbrtf(XYZ.y);
    else
      cubeRootY = (T2 * XYZ.y + 16) / 116;
    
    if (XYZ.z > T1)
      cubeRootZ = cbrtf(XYZ.z);
    else
      cubeRootZ = (T2 * XYZ.z + 16) / 116;
    
    
    
    dst[c] = make_float4(116*cubeRootY-16, 500*(cubeRootX-cubeRootY), 200*(cubeRootY-cubeRootZ), in.w);
  }
}


//-----------------------------------------------------------------------------
/** convert kernel lab -> rgb
 */
__global__ void cuConvertLABToRGBKernel(const float4* src, float4* dst, size_t stride,
                                        int width, int height)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int c = y*stride + x;

  if (x<width && y<height)
  {
    // Read
    float4 in = src[c];
    
    float L = in.x;
    float a = in.y;
    float b = in.z;

    
    // convert to XYZ
    const float T1 = cbrtf(216/24389.0f);
    const float fy = (L+16) / 116.0f;
    
    float4 XYZ;
    if (L > 8)
      XYZ.y = fy*fy*fy;
    else
      XYZ.y = L / (24389/27.0f);
    
    float fx = a/500.0f + fy;
    if (fx > T1)
      XYZ.x = fx*fx*fx;
    else
      XYZ.x = (116*fx-16) / (24389/27.0f);
    
    float fz = fy - b/200.0f;
    if (fz > T1)
      XYZ.z = fz*fz*fz;
    else
      XYZ.z = (116*fz-16) / (24389/27.0f);
    
    
    // Normalize for D65 white point
    XYZ.x *= 0.950456f;
    XYZ.z *= 1.088754f;
    
    float4 rgb;
    rgb.x = 3.2404542f*XYZ.x + -1.5371385f*XYZ.y + -0.4985314f*XYZ.z;
    rgb.y = -0.9692660f*XYZ.x + 1.8760108f*XYZ.y + 0.0415560f*XYZ.z;
    rgb.z = 0.0556434f*XYZ.x + -0.2040259f*XYZ.y + 1.0572252f*XYZ.z;
    rgb.w = in.w;
    
    dst[c] = rgb;
  }
}



/* ***************************************************************************
 *  CUDA WRAPPERS
 * ***************************************************************************/

//-----------------------------------------------------------------------------
/** convert kernel 32f_C3 -> 32f_C4 (float3 -> float4)
 */
void cuConvert(const iu::ImageGpu_32f_C3* src,
                   iu::ImageGpu_32f_C4* dst)
{
    assert(src->size() == dst->size());
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(src->width(), dimBlock.x),
               iu::divUp(src->height(), dimBlock.y));

  cuConvertC3ToC4Kernel<<<dimGrid, dimBlock>>>(src->data(), src->stride(),
                                               dst->data(), dst->stride(), src->width(), src->height());

}

//-----------------------------------------------------------------------------
/** convert kernel 32f_C4 -> 32f_C3 (float4 -> float3)
 */
void cuConvert(const iu::ImageGpu_32f_C4* src,
                   iu::ImageGpu_32f_C3* dst)
{
    assert(src->size() == dst->size());
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(src->width(), dimBlock.x),
               iu::divUp(src->height(), dimBlock.y));

  cuConvertC4ToC3Kernel<<<dimGrid, dimBlock>>>(src->data(), src->stride(),
                                               dst->data(), dst->stride(), src->width(), src->height());

}


//-----------------------------------------------------------------------------
void cuConvert_8u_32f(const iu::ImageGpu_8u_C1* src,
                          iu::ImageGpu_32f_C1* dst, float mul_constant,
                          float add_constant)
{
    assert(src->size() == dst->size());
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(src->width(), dimBlock.x),
               iu::divUp(src->height(), dimBlock.y));

  cuConvert8uC1To32fC1Kernel<<<dimGrid, dimBlock>>>(src->data(), src->stride(), dst->data(), dst->stride(),
                                                    mul_constant, add_constant, src->width(), src->height());

}

void cuConvert_32u_32f(const iu::ImageGpu_32u_C1* src,
                                 iu::ImageGpu_32f_C1* dst,
                                 float mul_constant,  float add_constant)
{
    assert(src->size() == dst->size());
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(src->width(), dimBlock.x),
               iu::divUp(src->height(), dimBlock.y));

  cuConvert32uC1To32fC1Kernel<<<dimGrid, dimBlock>>>(src->data(), src->stride(), dst->data(), dst->stride(),
                                                    mul_constant, add_constant, src->width(), src->height());
}

//-----------------------------------------------------------------------------
struct int2float
{
	__host__ __device__ float operator()(int x) const
	{
		return (float)x;
	}
};

void cuConvert_32s_32f_lin(iu::LinearDeviceMemory_32s_C1 *src, iu::LinearDeviceMemory_32f_C1 *dest)
{
	thrust::transform(src->begin(), src->end(), dest->begin(), int2float());
}

//-----------------------------------------------------------------------------
void cuConvert_8u_32f_C3C4(const iu::ImageGpu_8u_C3* src,
                          iu::ImageGpu_32f_C4* dst, float mul_constant,
                          float add_constant)
{
    assert(src->size() == dst->size());
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(src->width(), dimBlock.x),
               iu::divUp(src->height(), dimBlock.y));
  
  cuConvert8uC3To32fC4Kernel<<<dimGrid, dimBlock>>>(&(src->data()->x), src->pitch(), dst->data(), dst->stride(),
                                                    mul_constant, add_constant, src->width(), src->height());

}


//-----------------------------------------------------------------------------
void cuConvert_32f_8u(const iu::ImageGpu_32f_C1* src,
                          iu::ImageGpu_8u_C1* dst, float mul_constant,
                          unsigned char add_constant)
{
    assert(src->size() == dst->size());
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(src->width(), dimBlock.x),
               iu::divUp(src->height(), dimBlock.y));

  cuConvert32fC1To8uC1Kernel<<<dimGrid, dimBlock>>>(src->data(), src->stride(), dst->data(), dst->stride(),
                                                    mul_constant, add_constant, src->width(), src->height());

}

//-----------------------------------------------------------------------------
void cuConvert_32f_8u(const iu::ImageGpu_32f_C4* src,
                          iu::ImageGpu_8u_C4* dst, float mul_constant,
                          unsigned char add_constant)
{
    assert(src->size() == dst->size());
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(src->width(), dimBlock.x),
               iu::divUp(src->height(), dimBlock.y));

  cuConvert32fC4To8uC4Kernel<<<dimGrid, dimBlock>>>(src->data(), src->stride(), dst->data(), dst->stride(),
                                                    mul_constant, add_constant, src->width(), src->height());

}

//-----------------------------------------------------------------------------
void cuConvert_rgb_to_hsv(const iu::ImageGpu_32f_C4* src, iu::ImageGpu_32f_C4* dst,
                              bool normalize)
{
    assert(src->size() == dst->size());
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(src->width(), dimBlock.x),
               iu::divUp(src->height(), dimBlock.y));

  cuConvertRGBToHSVKernel<<<dimGrid, dimBlock>>>(src->data(), dst->data(), src->stride(),
                                                 src->width(), src->height(), normalize);

}

//-----------------------------------------------------------------------------
void cuConvert_hsv_to_rgb(const iu::ImageGpu_32f_C4* src, iu::ImageGpu_32f_C4* dst,
                              bool denormalize)
{
    assert(src->size() == dst->size());
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(src->width(), dimBlock.x),
               iu::divUp(src->height(), dimBlock.y));

  cuConvertHSVToRGBKernel<<<dimGrid, dimBlock>>>(src->data(), dst->data(), src->stride(),
                                                 src->width(), src->height(), denormalize);

}


//-----------------------------------------------------------------------------
void cuConvert_rgb_to_lab(const iu::ImageGpu_32f_C4* src, iu::ImageGpu_32f_C4* dst, bool isNormalized)
{
    assert(src->size() == dst->size());
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(src->width(), dimBlock.x),
               iu::divUp(src->height(), dimBlock.y));

  cuConvertRGBToLABKernel<<<dimGrid, dimBlock>>>(src->data(), dst->data(), src->stride(),
                                                 src->width(), src->height(), isNormalized);

}

//-----------------------------------------------------------------------------
void cuConvert_lab_to_rgb(const iu::ImageGpu_32f_C4* src, iu::ImageGpu_32f_C4* dst)
{
    assert(src->size() == dst->size());
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(src->width(), dimBlock.x),
               iu::divUp(src->height(), dimBlock.y));

  cuConvertLABToRGBKernel<<<dimGrid, dimBlock>>>(src->data(), dst->data(), src->stride(),
                                                 src->width(), src->height());

}





//typedef float num_t;

//__global__ void sum_reduce(iu::ImageGpu_32f_C1::KernelData g_data, num_t* sum)
//{
//    const unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
//    const unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
//    const int linId = threadIdx.x + threadIdx.y*blockDim.x;

//    extern volatile __shared__ num_t reduction[];

//    if (x<g_data.width_ && y<g_data.height_)
//    {
//        reduction[linId] = g_data(x,y);
//    }
//    else
//        reduction[linId] = 0;

//    __syncthreads();

//    if (linId < 128) { reduction[linId] += reduction[linId+128]; }
//    __syncthreads();
//    if (linId < 64) { reduction[linId] += reduction[linId+64]; }
//    __syncthreads();

//    if (linId < 32)
//    {
//        reduction[linId] += reduction[linId+32];
//        reduction[linId] += reduction[linId+16];
//        reduction[linId] += reduction[linId+8];
//        reduction[linId] += reduction[linId+4];
//        reduction[linId] += reduction[linId+2];
//        reduction[linId] += reduction[linId+1];
//    }
//    __syncthreads();



//    if (linId == 0)
//        atomicAdd(sum, reduction[0]);
//}


//double cuSummation(iu::ImageGpu_32f_C1* src)
//{
//    // fragmentation
//    unsigned int block_size = 16;
//    dim3 dimBlock(block_size, block_size);
//    dim3 dimGrid(iu::divUp(src->width(), dimBlock.x), iu::divUp(src->height(), dimBlock.y));

//    num_t* s;
//    hipMalloc(&s, sizeof(num_t));
//    hipMemset(s, 0, sizeof(num_t));

//    int shared_sz = block_size*block_size*sizeof(num_t);
//    sum_reduce<<< dimGrid, dimBlock, shared_sz >>> (*src, s);

//    num_t sum = 0;
//    hipMemcpy(&sum, s, sizeof(num_t), hipMemcpyDeviceToHost);

//    hipFree(s);
//    return sum;
//}


} // namespace iuprivate


