#include "hip/hip_runtime.h"

#include "iumath.h"
#include "iucore.h"
#include "iumath/arithmetics.cuh"

namespace iu {
namespace math {

// add constant
void addC(iu::ImageGpu_32f_C1& src, const float& val, iu::ImageGpu_32f_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_32f_C2& src, const float2& val, iu::ImageGpu_32f_C2& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_32f_C3& src, const float3& val, iu::ImageGpu_32f_C3& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_32f_C4& src, const float4& val, iu::ImageGpu_32f_C4& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::ImageGpu_32s_C1& src, const int& val, iu::ImageGpu_32s_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_32u_C1& src, const unsigned int& val, iu::ImageGpu_32u_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_16u_C1& src, const unsigned short& val, iu::ImageGpu_16u_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_8u_C1& src, const unsigned char& val, iu::ImageGpu_8u_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_8u_C2& src, const uchar2& val, iu::ImageGpu_8u_C2& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_8u_C3& src, const uchar3& val, iu::ImageGpu_8u_C3& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_8u_C4& src, const uchar4& val, iu::ImageGpu_8u_C4& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::LinearDeviceMemory_32f_C1& src, const float& val, iu::LinearDeviceMemory_32f_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_32f_C2& src, const float2& val, iu::LinearDeviceMemory_32f_C2& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_32f_C3& src, const float3& val, iu::LinearDeviceMemory_32f_C3& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_32f_C4& src, const float4& val, iu::LinearDeviceMemory_32f_C4& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::LinearDeviceMemory_32s_C1& src, const int& val, iu::LinearDeviceMemory_32s_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_32u_C1& src, const unsigned int& val, iu::LinearDeviceMemory_32u_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_16u_C1& src, const unsigned short& val, iu::LinearDeviceMemory_16u_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_8u_C1& src, const unsigned char& val, iu::LinearDeviceMemory_8u_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_8u_C2& src, const uchar2& val, iu::LinearDeviceMemory_8u_C2& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_8u_C3& src, const uchar3& val, iu::LinearDeviceMemory_8u_C3& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_8u_C4& src, const uchar4& val, iu::LinearDeviceMemory_8u_C4& dst) {iuprivate::math::addC(src,val,dst);}

// multiply constant
void mulC(iu::ImageGpu_32f_C1& src, const float& val, iu::ImageGpu_32f_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_32f_C2& src, const float2& val, iu::ImageGpu_32f_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_32f_C3& src, const float3& val, iu::ImageGpu_32f_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_32f_C4& src, const float4& val, iu::ImageGpu_32f_C4& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::ImageGpu_32s_C1& src, const int& val, iu::ImageGpu_32s_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_32u_C1& src, const unsigned int& val, iu::ImageGpu_32u_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_16u_C1& src, const unsigned short& val, iu::ImageGpu_16u_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_8u_C1& src, const unsigned char& val, iu::ImageGpu_8u_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_8u_C2& src, const uchar2& val, iu::ImageGpu_8u_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_8u_C3& src, const uchar3& val, iu::ImageGpu_8u_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_8u_C4& src, const uchar4& val, iu::ImageGpu_8u_C4& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::LinearDeviceMemory_32f_C1& src, const float& val, iu::LinearDeviceMemory_32f_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_32f_C2& src, const float2& val, iu::LinearDeviceMemory_32f_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_32f_C3& src, const float3& val, iu::LinearDeviceMemory_32f_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_32f_C4& src, const float4& val, iu::LinearDeviceMemory_32f_C4& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::LinearDeviceMemory_32s_C1& src, const int& val, iu::LinearDeviceMemory_32s_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_32u_C1& src, const unsigned int& val, iu::LinearDeviceMemory_32u_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_16u_C1& src, const unsigned short& val, iu::LinearDeviceMemory_16u_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_8u_C1& src, const unsigned char& val, iu::LinearDeviceMemory_8u_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_8u_C2& src, const uchar2& val, iu::LinearDeviceMemory_8u_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_8u_C3& src, const uchar3& val, iu::LinearDeviceMemory_8u_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_8u_C4& src, const uchar4& val, iu::LinearDeviceMemory_8u_C4& dst) {iuprivate::math::mulC(src,val,dst);}

// pointwise weighted add
void addWeighted(iu::ImageGpu_32f_C1& src1, const float& weight1,
                 iu::ImageGpu_32f_C1& src2, const float& weight2,iu::ImageGpu_32f_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageGpu_32f_C2& src1, const float2& weight1,
                 iu::ImageGpu_32f_C2& src2, const float2& weight2,iu::ImageGpu_32f_C2& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageGpu_32f_C3& src1, const float3& weight1,
                 iu::ImageGpu_32f_C3& src2, const float3& weight2,iu::ImageGpu_32f_C3& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageGpu_32f_C4& src1, const float4& weight1,
                 iu::ImageGpu_32f_C4& src2, const float4& weight2,iu::ImageGpu_32f_C4& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageGpu_8u_C1& src1, const unsigned char& weight1,
                 iu::ImageGpu_8u_C1& src2, const unsigned char& weight2,iu::ImageGpu_8u_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageGpu_8u_C4& src1, const uchar4 &weight1,
                 iu::ImageGpu_8u_C4& src2, const uchar4& weight2, iu::ImageGpu_8u_C4& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}

void addWeighted(iu::LinearDeviceMemory_32f_C1& src1, const float& weight1,
                 iu::LinearDeviceMemory_32f_C1& src2, const float& weight2,iu::LinearDeviceMemory_32f_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearDeviceMemory_32f_C2& src1, const float2& weight1,
                 iu::LinearDeviceMemory_32f_C2& src2, const float2& weight2,iu::LinearDeviceMemory_32f_C2& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearDeviceMemory_32f_C3& src1, const float3& weight1,
                 iu::LinearDeviceMemory_32f_C3& src2, const float3& weight2,iu::LinearDeviceMemory_32f_C3& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearDeviceMemory_32f_C4& src1, const float4& weight1,
                 iu::LinearDeviceMemory_32f_C4& src2, const float4& weight2,iu::LinearDeviceMemory_32f_C4& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearDeviceMemory_8u_C1& src1, const unsigned char& weight1,
                 iu::LinearDeviceMemory_8u_C1& src2, const unsigned char& weight2,iu::LinearDeviceMemory_8u_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearDeviceMemory_8u_C4& src1, const uchar4 &weight1,
                 iu::LinearDeviceMemory_8u_C4& src2, const uchar4& weight2, iu::LinearDeviceMemory_8u_C4& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}

// pointwise multiply
void mul(iu::ImageGpu_32f_C1& src1, iu::ImageGpu_32f_C1& src2, iu::ImageGpu_32f_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::ImageGpu_32f_C2& src1, iu::ImageGpu_32f_C2& src2, iu::ImageGpu_32f_C2& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::ImageGpu_32f_C3& src1, iu::ImageGpu_32f_C3& src2, iu::ImageGpu_32f_C3& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::ImageGpu_32f_C4& src1, iu::ImageGpu_32f_C4& src2, iu::ImageGpu_32f_C4& dst) {iuprivate::math::mul(src1,src2,dst);}

void mul(iu::ImageGpu_8u_C1& src1, iu::ImageGpu_8u_C1& src2, iu::ImageGpu_8u_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::ImageGpu_8u_C4& src1, iu::ImageGpu_8u_C4& src2, iu::ImageGpu_8u_C4& dst) {iuprivate::math::mul(src1,src2,dst);}

void mul(iu::LinearDeviceMemory_32f_C1& src1, iu::LinearDeviceMemory_32f_C1& src2, iu::LinearDeviceMemory_32f_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::LinearDeviceMemory_32f_C2& src1, iu::LinearDeviceMemory_32f_C2& src2, iu::LinearDeviceMemory_32f_C2& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::LinearDeviceMemory_32f_C3& src1, iu::LinearDeviceMemory_32f_C3& src2, iu::LinearDeviceMemory_32f_C3& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::LinearDeviceMemory_32f_C4& src1, iu::LinearDeviceMemory_32f_C4& src2, iu::LinearDeviceMemory_32f_C4& dst) {iuprivate::math::mul(src1,src2,dst);}

void mul(iu::LinearDeviceMemory_8u_C1& src1, iu::LinearDeviceMemory_8u_C1& src2, iu::LinearDeviceMemory_8u_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::LinearDeviceMemory_8u_C4& src1, iu::LinearDeviceMemory_8u_C4& src2, iu::LinearDeviceMemory_8u_C4& dst) {iuprivate::math::mul(src1,src2,dst);}

} //namespace math
} //namespace iu
