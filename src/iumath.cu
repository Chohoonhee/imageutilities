#include "hip/hip_runtime.h"

#include "iumath.h"
#include "iucore.h"

#include "iumath/arithmetics.cuh"
#include "iumath/statistics.cuh"
#include "iumath/complex.cuh"
#include "iuhelpermath.h"
namespace iu {
namespace math {

// add constant
void addC(iu::ImageGpu_32f_C1& src, const float& val, iu::ImageGpu_32f_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_32f_C2& src, const float2& val, iu::ImageGpu_32f_C2& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_32f_C3& src, const float3& val, iu::ImageGpu_32f_C3& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_32f_C4& src, const float4& val, iu::ImageGpu_32f_C4& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::ImageGpu_32s_C1& src, const int& val, iu::ImageGpu_32s_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_32u_C1& src, const unsigned int& val, iu::ImageGpu_32u_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_16u_C1& src, const unsigned short& val, iu::ImageGpu_16u_C1& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::ImageGpu_8u_C1& src, const unsigned char& val, iu::ImageGpu_8u_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_8u_C2& src, const uchar2& val, iu::ImageGpu_8u_C2& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_8u_C3& src, const uchar3& val, iu::ImageGpu_8u_C3& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_8u_C4& src, const uchar4& val, iu::ImageGpu_8u_C4& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::VolumeGpu_32f_C1& src, const float& val, iu::VolumeGpu_32f_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::VolumeGpu_32f_C2& src, const float2& val, iu::VolumeGpu_32f_C2& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::LinearDeviceMemory_32f_C1& src, const float& val, iu::LinearDeviceMemory_32f_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_32f_C2& src, const float2& val, iu::LinearDeviceMemory_32f_C2& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_32f_C3& src, const float3& val, iu::LinearDeviceMemory_32f_C3& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_32f_C4& src, const float4& val, iu::LinearDeviceMemory_32f_C4& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::LinearDeviceMemory_32s_C1& src, const int& val, iu::LinearDeviceMemory_32s_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_32u_C1& src, const unsigned int& val, iu::LinearDeviceMemory_32u_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_16u_C1& src, const unsigned short& val, iu::LinearDeviceMemory_16u_C1& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::LinearDeviceMemory_8u_C1& src, const unsigned char& val, iu::LinearDeviceMemory_8u_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_8u_C2& src, const uchar2& val, iu::LinearDeviceMemory_8u_C2& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_8u_C3& src, const uchar3& val, iu::LinearDeviceMemory_8u_C3& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_8u_C4& src, const uchar4& val, iu::LinearDeviceMemory_8u_C4& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::ImageCpu_32f_C1& src, const float& val, iu::ImageCpu_32f_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageCpu_32f_C2& src, const float2& val, iu::ImageCpu_32f_C2& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageCpu_32f_C3& src, const float3& val, iu::ImageCpu_32f_C3& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageCpu_32f_C4& src, const float4& val, iu::ImageCpu_32f_C4& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::ImageCpu_32s_C1& src, const int& val, iu::ImageCpu_32s_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageCpu_32u_C1& src, const unsigned int& val, iu::ImageCpu_32u_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageCpu_16u_C1& src, const unsigned short& val, iu::ImageCpu_16u_C1& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::ImageCpu_8u_C1& src, const unsigned char& val, iu::ImageCpu_8u_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageCpu_8u_C2& src, const uchar2& val, iu::ImageCpu_8u_C2& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageCpu_8u_C3& src, const uchar3& val, iu::ImageCpu_8u_C3& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageCpu_8u_C4& src, const uchar4& val, iu::ImageCpu_8u_C4& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::VolumeCpu_32f_C1& src, const float& val, iu::VolumeCpu_32f_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::VolumeCpu_32f_C2& src, const float2& val, iu::VolumeCpu_32f_C2& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::LinearHostMemory_32f_C1& src, const float& val, iu::LinearHostMemory_32f_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearHostMemory_32f_C2& src, const float2& val, iu::LinearHostMemory_32f_C2& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearHostMemory_32f_C3& src, const float3& val, iu::LinearHostMemory_32f_C3& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearHostMemory_32f_C4& src, const float4& val, iu::LinearHostMemory_32f_C4& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::LinearHostMemory_32s_C1& src, const int& val, iu::LinearHostMemory_32s_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearHostMemory_32u_C1& src, const unsigned int& val, iu::LinearHostMemory_32u_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearHostMemory_16u_C1& src, const unsigned short& val, iu::LinearHostMemory_16u_C1& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::LinearHostMemory_8u_C1& src, const unsigned char& val, iu::LinearHostMemory_8u_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearHostMemory_8u_C2& src, const uchar2& val, iu::LinearHostMemory_8u_C2& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearHostMemory_8u_C3& src, const uchar3& val, iu::LinearHostMemory_8u_C3& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearHostMemory_8u_C4& src, const uchar4& val, iu::LinearHostMemory_8u_C4& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::ImageGpu_64f_C1& src, const double& val, iu::ImageGpu_64f_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_64f_C2& src, const double2& val, iu::ImageGpu_64f_C2& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_64f_C3& src, const double3& val, iu::ImageGpu_64f_C3& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_64f_C4& src, const double4& val, iu::ImageGpu_64f_C4& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::VolumeGpu_64f_C1& src, const double& val, iu::VolumeGpu_64f_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::VolumeGpu_64f_C2& src, const double2& val, iu::VolumeGpu_64f_C2& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::LinearDeviceMemory_64f_C1& src, const double& val, iu::LinearDeviceMemory_64f_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_64f_C2& src, const double2& val, iu::LinearDeviceMemory_64f_C2& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_64f_C3& src, const double3& val, iu::LinearDeviceMemory_64f_C3& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearDeviceMemory_64f_C4& src, const double4& val, iu::LinearDeviceMemory_64f_C4& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::ImageCpu_64f_C1& src, const double& val, iu::ImageCpu_64f_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageCpu_64f_C2& src, const double2& val, iu::ImageCpu_64f_C2& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageCpu_64f_C3& src, const double3& val, iu::ImageCpu_64f_C3& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageCpu_64f_C4& src, const double4& val, iu::ImageCpu_64f_C4& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::VolumeCpu_64f_C1& src, const double& val, iu::VolumeCpu_64f_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::VolumeCpu_64f_C2& src, const double2& val, iu::VolumeCpu_64f_C2& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::LinearHostMemory_64f_C1& src, const double& val, iu::LinearHostMemory_64f_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearHostMemory_64f_C2& src, const double2& val, iu::LinearHostMemory_64f_C2& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearHostMemory_64f_C3& src, const double3& val, iu::LinearHostMemory_64f_C3& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::LinearHostMemory_64f_C4& src, const double4& val, iu::LinearHostMemory_64f_C4& dst) {iuprivate::math::addC(src,val,dst);}

// multiply constant
void mulC(iu::ImageGpu_32f_C1& src, const float& val, iu::ImageGpu_32f_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_32f_C2& src, const float2& val, iu::ImageGpu_32f_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_32f_C3& src, const float3& val, iu::ImageGpu_32f_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_32f_C4& src, const float4& val, iu::ImageGpu_32f_C4& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::ImageGpu_32s_C1& src, const int& val, iu::ImageGpu_32s_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_32u_C1& src, const unsigned int& val, iu::ImageGpu_32u_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_16u_C1& src, const unsigned short& val, iu::ImageGpu_16u_C1& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::ImageGpu_8u_C1& src, const unsigned char& val, iu::ImageGpu_8u_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_8u_C2& src, const uchar2& val, iu::ImageGpu_8u_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_8u_C3& src, const uchar3& val, iu::ImageGpu_8u_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_8u_C4& src, const uchar4& val, iu::ImageGpu_8u_C4& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::VolumeGpu_32f_C1& src, const float& val, iu::VolumeGpu_32f_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::VolumeGpu_32f_C2& src, const float2& val, iu::VolumeGpu_32f_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::VolumeGpu_32f_C3& src, const float3& val, iu::VolumeGpu_32f_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::VolumeGpu_32f_C4& src, const float4& val, iu::VolumeGpu_32f_C4& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::LinearDeviceMemory_32f_C1& src, const float& val, iu::LinearDeviceMemory_32f_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_32f_C2& src, const float2& val, iu::LinearDeviceMemory_32f_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_32f_C3& src, const float3& val, iu::LinearDeviceMemory_32f_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_32f_C4& src, const float4& val, iu::LinearDeviceMemory_32f_C4& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::LinearDeviceMemory_32s_C1& src, const int& val, iu::LinearDeviceMemory_32s_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_32u_C1& src, const unsigned int& val, iu::LinearDeviceMemory_32u_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_16u_C1& src, const unsigned short& val, iu::LinearDeviceMemory_16u_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_8u_C1& src, const unsigned char& val, iu::LinearDeviceMemory_8u_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_8u_C2& src, const uchar2& val, iu::LinearDeviceMemory_8u_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_8u_C3& src, const uchar3& val, iu::LinearDeviceMemory_8u_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_8u_C4& src, const uchar4& val, iu::LinearDeviceMemory_8u_C4& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::ImageCpu_32f_C1& src, const float& val, iu::ImageCpu_32f_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageCpu_32f_C2& src, const float2& val, iu::ImageCpu_32f_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageCpu_32f_C3& src, const float3& val, iu::ImageCpu_32f_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageCpu_32f_C4& src, const float4& val, iu::ImageCpu_32f_C4& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::ImageCpu_32s_C1& src, const int& val, iu::ImageCpu_32s_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageCpu_32u_C1& src, const unsigned int& val, iu::ImageCpu_32u_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageCpu_16u_C1& src, const unsigned short& val, iu::ImageCpu_16u_C1& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::ImageCpu_8u_C1& src, const unsigned char& val, iu::ImageCpu_8u_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageCpu_8u_C2& src, const uchar2& val, iu::ImageCpu_8u_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageCpu_8u_C3& src, const uchar3& val, iu::ImageCpu_8u_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageCpu_8u_C4& src, const uchar4& val, iu::ImageCpu_8u_C4& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::VolumeCpu_32f_C1& src, const float& val, iu::VolumeCpu_32f_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::VolumeCpu_32f_C2& src, const float2& val, iu::VolumeCpu_32f_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::VolumeCpu_32f_C3& src, const float3& val, iu::VolumeCpu_32f_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::VolumeCpu_32f_C4& src, const float4& val, iu::VolumeCpu_32f_C4& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::LinearHostMemory_32f_C1& src, const float& val, iu::LinearHostMemory_32f_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearHostMemory_32f_C2& src, const float2& val, iu::LinearHostMemory_32f_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearHostMemory_32f_C3& src, const float3& val, iu::LinearHostMemory_32f_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearHostMemory_32f_C4& src, const float4& val, iu::LinearHostMemory_32f_C4& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::LinearHostMemory_32s_C1& src, const int& val, iu::LinearHostMemory_32s_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearHostMemory_32u_C1& src, const unsigned int& val, iu::LinearHostMemory_32u_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearHostMemory_16u_C1& src, const unsigned short& val, iu::LinearHostMemory_16u_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearHostMemory_8u_C1& src, const unsigned char& val, iu::LinearHostMemory_8u_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearHostMemory_8u_C2& src, const uchar2& val, iu::LinearHostMemory_8u_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearHostMemory_8u_C3& src, const uchar3& val, iu::LinearHostMemory_8u_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearHostMemory_8u_C4& src, const uchar4& val, iu::LinearHostMemory_8u_C4& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::ImageGpu_64f_C1& src, const double& val, iu::ImageGpu_64f_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_64f_C2& src, const double2& val, iu::ImageGpu_64f_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_64f_C3& src, const double3& val, iu::ImageGpu_64f_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_64f_C4& src, const double4& val, iu::ImageGpu_64f_C4& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::VolumeGpu_64f_C1& src, const double& val, iu::VolumeGpu_64f_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::VolumeGpu_64f_C2& src, const double2& val, iu::VolumeGpu_64f_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::VolumeGpu_64f_C3& src, const double3& val, iu::VolumeGpu_64f_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::VolumeGpu_64f_C4& src, const double4& val, iu::VolumeGpu_64f_C4& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::LinearDeviceMemory_64f_C1& src, const double& val, iu::LinearDeviceMemory_64f_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_64f_C2& src, const double2& val, iu::LinearDeviceMemory_64f_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_64f_C3& src, const double3& val, iu::LinearDeviceMemory_64f_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearDeviceMemory_64f_C4& src, const double4& val, iu::LinearDeviceMemory_64f_C4& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::ImageCpu_64f_C1& src, const double& val, iu::ImageCpu_64f_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageCpu_64f_C2& src, const double2& val, iu::ImageCpu_64f_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageCpu_64f_C3& src, const double3& val, iu::ImageCpu_64f_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageCpu_64f_C4& src, const double4& val, iu::ImageCpu_64f_C4& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::VolumeCpu_64f_C1& src, const double& val, iu::VolumeCpu_64f_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::VolumeCpu_64f_C2& src, const double2& val, iu::VolumeCpu_64f_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::VolumeCpu_64f_C3& src, const double3& val, iu::VolumeCpu_64f_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::VolumeCpu_64f_C4& src, const double4& val, iu::VolumeCpu_64f_C4& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::LinearHostMemory_64f_C1& src, const double& val, iu::LinearHostMemory_64f_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearHostMemory_64f_C2& src, const double2& val, iu::LinearHostMemory_64f_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearHostMemory_64f_C3& src, const double3& val, iu::LinearHostMemory_64f_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::LinearHostMemory_64f_C4& src, const double4& val, iu::LinearHostMemory_64f_C4& dst) {iuprivate::math::mulC(src,val,dst);}

// pointwise weighted add
void addWeighted(iu::ImageGpu_32f_C1& src1, const float& weight1,
                 iu::ImageGpu_32f_C1& src2, const float& weight2,iu::ImageGpu_32f_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageGpu_32f_C2& src1, const float2& weight1,
                 iu::ImageGpu_32f_C2& src2, const float2& weight2,iu::ImageGpu_32f_C2& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageGpu_32f_C3& src1, const float3& weight1,
                 iu::ImageGpu_32f_C3& src2, const float3& weight2,iu::ImageGpu_32f_C3& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageGpu_32f_C4& src1, const float4& weight1,
                 iu::ImageGpu_32f_C4& src2, const float4& weight2,iu::ImageGpu_32f_C4& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageGpu_8u_C1& src1, const unsigned char& weight1,
                 iu::ImageGpu_8u_C1& src2, const unsigned char& weight2,iu::ImageGpu_8u_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageGpu_8u_C4& src1, const uchar4 &weight1,
                 iu::ImageGpu_8u_C4& src2, const uchar4& weight2, iu::ImageGpu_8u_C4& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}

void addWeighted(iu::VolumeGpu_32f_C1& src1, const float& weight1,
                 iu::VolumeGpu_32f_C1& src2, const float& weight2,iu::VolumeGpu_32f_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}

void addWeighted(iu::LinearDeviceMemory_32f_C1& src1, const float& weight1,
                 iu::LinearDeviceMemory_32f_C1& src2, const float& weight2,iu::LinearDeviceMemory_32f_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearDeviceMemory_32f_C2& src1, const float2& weight1,
                 iu::LinearDeviceMemory_32f_C2& src2, const float2& weight2,iu::LinearDeviceMemory_32f_C2& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearDeviceMemory_32f_C3& src1, const float3& weight1,
                 iu::LinearDeviceMemory_32f_C3& src2, const float3& weight2,iu::LinearDeviceMemory_32f_C3& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearDeviceMemory_32f_C4& src1, const float4& weight1,
                 iu::LinearDeviceMemory_32f_C4& src2, const float4& weight2,iu::LinearDeviceMemory_32f_C4& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearDeviceMemory_8u_C1& src1, const unsigned char& weight1,
                 iu::LinearDeviceMemory_8u_C1& src2, const unsigned char& weight2,iu::LinearDeviceMemory_8u_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearDeviceMemory_8u_C4& src1, const uchar4 &weight1,
                 iu::LinearDeviceMemory_8u_C4& src2, const uchar4& weight2, iu::LinearDeviceMemory_8u_C4& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}

void addWeighted(iu::ImageCpu_32f_C1& src1, const float& weight1,
                 iu::ImageCpu_32f_C1& src2, const float& weight2,iu::ImageCpu_32f_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageCpu_32f_C2& src1, const float2& weight1,
                 iu::ImageCpu_32f_C2& src2, const float2& weight2,iu::ImageCpu_32f_C2& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageCpu_32f_C3& src1, const float3& weight1,
                 iu::ImageCpu_32f_C3& src2, const float3& weight2,iu::ImageCpu_32f_C3& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageCpu_32f_C4& src1, const float4& weight1,
                 iu::ImageCpu_32f_C4& src2, const float4& weight2,iu::ImageCpu_32f_C4& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageCpu_8u_C1& src1, const unsigned char& weight1,
                 iu::ImageCpu_8u_C1& src2, const unsigned char& weight2,iu::ImageCpu_8u_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageCpu_8u_C4& src1, const uchar4 &weight1,
                 iu::ImageCpu_8u_C4& src2, const uchar4& weight2, iu::ImageCpu_8u_C4& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}

void addWeighted(iu::VolumeCpu_32f_C1& src1, const float& weight1,
                 iu::VolumeCpu_32f_C1& src2, const float& weight2,iu::VolumeCpu_32f_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}

void addWeighted(iu::LinearHostMemory_32f_C1& src1, const float& weight1,
                 iu::LinearHostMemory_32f_C1& src2, const float& weight2,iu::LinearHostMemory_32f_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearHostMemory_32f_C2& src1, const float2& weight1,
                 iu::LinearHostMemory_32f_C2& src2, const float2& weight2,iu::LinearHostMemory_32f_C2& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearHostMemory_32f_C3& src1, const float3& weight1,
                 iu::LinearHostMemory_32f_C3& src2, const float3& weight2,iu::LinearHostMemory_32f_C3& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearHostMemory_32f_C4& src1, const float4& weight1,
                 iu::LinearHostMemory_32f_C4& src2, const float4& weight2,iu::LinearHostMemory_32f_C4& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearHostMemory_8u_C1& src1, const unsigned char& weight1,
                 iu::LinearHostMemory_8u_C1& src2, const unsigned char& weight2,iu::LinearHostMemory_8u_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearHostMemory_8u_C4& src1, const uchar4 &weight1,
                 iu::LinearHostMemory_8u_C4& src2, const uchar4& weight2, iu::LinearHostMemory_8u_C4& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}

void addWeighted(iu::ImageGpu_64f_C1& src1, const double& weight1,
                 iu::ImageGpu_64f_C1& src2, const double& weight2,iu::ImageGpu_64f_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageGpu_64f_C2& src1, const double2& weight1,
                 iu::ImageGpu_64f_C2& src2, const double2& weight2,iu::ImageGpu_64f_C2& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageGpu_64f_C3& src1, const double3& weight1,
                 iu::ImageGpu_64f_C3& src2, const double3& weight2,iu::ImageGpu_64f_C3& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageGpu_64f_C4& src1, const double4& weight1,
                 iu::ImageGpu_64f_C4& src2, const double4& weight2,iu::ImageGpu_64f_C4& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}

void addWeighted(iu::VolumeGpu_64f_C1& src1, const double& weight1,
                 iu::VolumeGpu_64f_C1& src2, const double& weight2,iu::VolumeGpu_64f_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}

void addWeighted(iu::LinearDeviceMemory_64f_C1& src1, const double& weight1,
                 iu::LinearDeviceMemory_64f_C1& src2, const double& weight2,iu::LinearDeviceMemory_64f_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearDeviceMemory_64f_C2& src1, const double2& weight1,
                 iu::LinearDeviceMemory_64f_C2& src2, const double2& weight2,iu::LinearDeviceMemory_64f_C2& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearDeviceMemory_64f_C3& src1, const double3& weight1,
                 iu::LinearDeviceMemory_64f_C3& src2, const double3& weight2,iu::LinearDeviceMemory_64f_C3& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearDeviceMemory_64f_C4& src1, const double4& weight1,
                 iu::LinearDeviceMemory_64f_C4& src2, const double4& weight2,iu::LinearDeviceMemory_64f_C4& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}

void addWeighted(iu::ImageCpu_64f_C1& src1, const double& weight1,
                 iu::ImageCpu_64f_C1& src2, const double& weight2,iu::ImageCpu_64f_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageCpu_64f_C2& src1, const double2& weight1,
                 iu::ImageCpu_64f_C2& src2, const double2& weight2,iu::ImageCpu_64f_C2& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageCpu_64f_C3& src1, const double3& weight1,
                 iu::ImageCpu_64f_C3& src2, const double3& weight2,iu::ImageCpu_64f_C3& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageCpu_64f_C4& src1, const double4& weight1,
                 iu::ImageCpu_64f_C4& src2, const double4& weight2,iu::ImageCpu_64f_C4& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}

void addWeighted(iu::VolumeCpu_64f_C1& src1, const double& weight1,
                 iu::VolumeCpu_64f_C1& src2, const double& weight2,iu::VolumeCpu_64f_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}

void addWeighted(iu::LinearHostMemory_64f_C1& src1, const double& weight1,
                 iu::LinearHostMemory_64f_C1& src2, const double& weight2,iu::LinearHostMemory_64f_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearHostMemory_64f_C2& src1, const double2& weight1,
                 iu::LinearHostMemory_64f_C2& src2, const double2& weight2,iu::LinearHostMemory_64f_C2& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearHostMemory_64f_C3& src1, const double3& weight1,
                 iu::LinearHostMemory_64f_C3& src2, const double3& weight2,iu::LinearHostMemory_64f_C3& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::LinearHostMemory_64f_C4& src1, const double4& weight1,
                 iu::LinearHostMemory_64f_C4& src2, const double4& weight2,iu::LinearHostMemory_64f_C4& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}

// pointwise multiply
void mul(iu::ImageGpu_32f_C1& src1, iu::ImageGpu_32f_C1& src2, iu::ImageGpu_32f_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::ImageGpu_32f_C2& src1, iu::ImageGpu_32f_C2& src2, iu::ImageGpu_32f_C2& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::ImageGpu_32f_C3& src1, iu::ImageGpu_32f_C3& src2, iu::ImageGpu_32f_C3& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::ImageGpu_32f_C4& src1, iu::ImageGpu_32f_C4& src2, iu::ImageGpu_32f_C4& dst) {iuprivate::math::mul(src1,src2,dst);}

void mul(iu::ImageGpu_8u_C1& src1, iu::ImageGpu_8u_C1& src2, iu::ImageGpu_8u_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::ImageGpu_8u_C4& src1, iu::ImageGpu_8u_C4& src2, iu::ImageGpu_8u_C4& dst) {iuprivate::math::mul(src1,src2,dst);}

void mul(iu::VolumeGpu_32f_C1& src1, iu::VolumeGpu_32f_C1& src2, iu::VolumeGpu_32f_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::VolumeGpu_32f_C2& src1, iu::VolumeGpu_32f_C2& src2, iu::VolumeGpu_32f_C2& dst) {iuprivate::math::mul(src1,src2,dst);}

void mul(iu::LinearDeviceMemory_32f_C1& src1, iu::LinearDeviceMemory_32f_C1& src2, iu::LinearDeviceMemory_32f_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::LinearDeviceMemory_32f_C2& src1, iu::LinearDeviceMemory_32f_C2& src2, iu::LinearDeviceMemory_32f_C2& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::LinearDeviceMemory_32f_C3& src1, iu::LinearDeviceMemory_32f_C3& src2, iu::LinearDeviceMemory_32f_C3& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::LinearDeviceMemory_32f_C4& src1, iu::LinearDeviceMemory_32f_C4& src2, iu::LinearDeviceMemory_32f_C4& dst) {iuprivate::math::mul(src1,src2,dst);}

void mul(iu::LinearDeviceMemory_8u_C1& src1, iu::LinearDeviceMemory_8u_C1& src2, iu::LinearDeviceMemory_8u_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::LinearDeviceMemory_8u_C4& src1, iu::LinearDeviceMemory_8u_C4& src2, iu::LinearDeviceMemory_8u_C4& dst) {iuprivate::math::mul(src1,src2,dst);}

void mul(iu::ImageCpu_32f_C1& src1, iu::ImageCpu_32f_C1& src2, iu::ImageCpu_32f_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::ImageCpu_32f_C2& src1, iu::ImageCpu_32f_C2& src2, iu::ImageCpu_32f_C2& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::ImageCpu_32f_C3& src1, iu::ImageCpu_32f_C3& src2, iu::ImageCpu_32f_C3& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::ImageCpu_32f_C4& src1, iu::ImageCpu_32f_C4& src2, iu::ImageCpu_32f_C4& dst) {iuprivate::math::mul(src1,src2,dst);}

void mul(iu::ImageCpu_8u_C1& src1, iu::ImageCpu_8u_C1& src2, iu::ImageCpu_8u_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::ImageCpu_8u_C4& src1, iu::ImageCpu_8u_C4& src2, iu::ImageCpu_8u_C4& dst) {iuprivate::math::mul(src1,src2,dst);}

void mul(iu::VolumeCpu_32f_C1& src1, iu::VolumeCpu_32f_C1& src2, iu::VolumeCpu_32f_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::VolumeCpu_32f_C2& src1, iu::VolumeCpu_32f_C2& src2, iu::VolumeCpu_32f_C2& dst) {iuprivate::math::mul(src1,src2,dst);}

void mul(iu::LinearHostMemory_32f_C1& src1, iu::LinearHostMemory_32f_C1& src2, iu::LinearHostMemory_32f_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::LinearHostMemory_32f_C2& src1, iu::LinearHostMemory_32f_C2& src2, iu::LinearHostMemory_32f_C2& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::LinearHostMemory_32f_C3& src1, iu::LinearHostMemory_32f_C3& src2, iu::LinearHostMemory_32f_C3& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::LinearHostMemory_32f_C4& src1, iu::LinearHostMemory_32f_C4& src2, iu::LinearHostMemory_32f_C4& dst) {iuprivate::math::mul(src1,src2,dst);}

void mul(iu::LinearHostMemory_8u_C1& src1, iu::LinearHostMemory_8u_C1& src2, iu::LinearHostMemory_8u_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::LinearHostMemory_8u_C4& src1, iu::LinearHostMemory_8u_C4& src2, iu::LinearHostMemory_8u_C4& dst) {iuprivate::math::mul(src1,src2,dst);}

void mul(iu::ImageGpu_64f_C1& src1, iu::ImageGpu_64f_C1& src2, iu::ImageGpu_64f_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::ImageGpu_64f_C2& src1, iu::ImageGpu_64f_C2& src2, iu::ImageGpu_64f_C2& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::ImageGpu_64f_C3& src1, iu::ImageGpu_64f_C3& src2, iu::ImageGpu_64f_C3& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::ImageGpu_64f_C4& src1, iu::ImageGpu_64f_C4& src2, iu::ImageGpu_64f_C4& dst) {iuprivate::math::mul(src1,src2,dst);}

void mul(iu::VolumeGpu_64f_C1& src1, iu::VolumeGpu_64f_C1& src2, iu::VolumeGpu_64f_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::VolumeGpu_64f_C2& src1, iu::VolumeGpu_64f_C2& src2, iu::VolumeGpu_64f_C2& dst) {iuprivate::math::mul(src1,src2,dst);}

void mul(iu::LinearDeviceMemory_64f_C1& src1, iu::LinearDeviceMemory_64f_C1& src2, iu::LinearDeviceMemory_64f_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::LinearDeviceMemory_64f_C2& src1, iu::LinearDeviceMemory_64f_C2& src2, iu::LinearDeviceMemory_64f_C2& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::LinearDeviceMemory_64f_C3& src1, iu::LinearDeviceMemory_64f_C3& src2, iu::LinearDeviceMemory_64f_C3& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::LinearDeviceMemory_64f_C4& src1, iu::LinearDeviceMemory_64f_C4& src2, iu::LinearDeviceMemory_64f_C4& dst) {iuprivate::math::mul(src1,src2,dst);}

void mul(iu::ImageCpu_64f_C1& src1, iu::ImageCpu_64f_C1& src2, iu::ImageCpu_64f_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::ImageCpu_64f_C2& src1, iu::ImageCpu_64f_C2& src2, iu::ImageCpu_64f_C2& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::ImageCpu_64f_C3& src1, iu::ImageCpu_64f_C3& src2, iu::ImageCpu_64f_C3& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::ImageCpu_64f_C4& src1, iu::ImageCpu_64f_C4& src2, iu::ImageCpu_64f_C4& dst) {iuprivate::math::mul(src1,src2,dst);}

void mul(iu::VolumeCpu_64f_C1& src1, iu::VolumeCpu_64f_C1& src2, iu::VolumeCpu_64f_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::VolumeCpu_64f_C2& src1, iu::VolumeCpu_64f_C2& src2, iu::VolumeCpu_64f_C2& dst) {iuprivate::math::mul(src1,src2,dst);}

void mul(iu::LinearHostMemory_64f_C1& src1, iu::LinearHostMemory_64f_C1& src2, iu::LinearHostMemory_64f_C1& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::LinearHostMemory_64f_C2& src1, iu::LinearHostMemory_64f_C2& src2, iu::LinearHostMemory_64f_C2& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::LinearHostMemory_64f_C3& src1, iu::LinearHostMemory_64f_C3& src2, iu::LinearHostMemory_64f_C3& dst) {iuprivate::math::mul(src1,src2,dst);}
void mul(iu::LinearHostMemory_64f_C4& src1, iu::LinearHostMemory_64f_C4& src2, iu::LinearHostMemory_64f_C4& dst) {iuprivate::math::mul(src1,src2,dst);}

// set value
void fill(iu::ImageGpu_32f_C1& dst, float value) {iuprivate::math::fill(dst,value);}
void fill(iu::ImageGpu_32f_C2& dst, float2 value) {iuprivate::math::fill(dst,value);}
void fill(iu::ImageGpu_32f_C4& dst, float4 value) {iuprivate::math::fill(dst,value);}
void fill(iu::ImageGpu_32u_C1& dst, unsigned int value) {iuprivate::math::fill(dst,value);}
void fill(iu::ImageGpu_8u_C1& dst, unsigned char value) {iuprivate::math::fill(dst,value);}
void fill(iu::ImageGpu_8u_C2& dst, uchar2 value) {iuprivate::math::fill(dst,value);}
void fill(iu::ImageGpu_8u_C4& dst, uchar4 value) {iuprivate::math::fill(dst,value);}

void fill(iu::ImageCpu_32f_C1& dst, float value) {iuprivate::math::fill(dst,value);}
void fill(iu::ImageCpu_32f_C2& dst, float2 value) {iuprivate::math::fill(dst,value);}
void fill(iu::ImageCpu_32f_C4& dst, float4 value) {iuprivate::math::fill(dst,value);}
void fill(iu::ImageCpu_8u_C1& dst, unsigned char value) {iuprivate::math::fill(dst,value);}
void fill(iu::ImageCpu_8u_C2& dst, uchar2 value) {iuprivate::math::fill(dst,value);}
void fill(iu::ImageCpu_8u_C4& dst, uchar4 value) {iuprivate::math::fill(dst,value);}

void fill(iu::LinearDeviceMemory_32f_C1& dst, float value) {iuprivate::math::fill(dst,value);}
void fill(iu::LinearDeviceMemory_32f_C2& dst, float2 value) {iuprivate::math::fill(dst,value);}
void fill(iu::LinearDeviceMemory_32f_C3& dst, float3 value) {iuprivate::math::fill(dst,value);}

void fill(iu::LinearHostMemory_32f_C1& dst, float value) {iuprivate::math::fill(dst,value);}
void fill(iu::LinearHostMemory_32f_C2& dst, float2 value) {iuprivate::math::fill(dst,value);}
void fill(iu::LinearHostMemory_32f_C3& dst, float3 value) {iuprivate::math::fill(dst,value);}

void fill(iu::VolumeGpu_32f_C1& dst, float value) {iuprivate::math::fill(dst,value);}
void fill(iu::VolumeGpu_32f_C2& dst, float2 value) {iuprivate::math::fill(dst,value);}
void fill(iu::VolumeGpu_32f_C3& dst, float3 value) {iuprivate::math::fill(dst,value);}

void fill(iu::VolumeCpu_32f_C1& dst, float value) {iuprivate::math::fill(dst,value);}
void fill(iu::VolumeCpu_32f_C2& dst, float2 value) {iuprivate::math::fill(dst,value);}
void fill(iu::VolumeCpu_32f_C3& dst, float3 value) {iuprivate::math::fill(dst,value);}

void fill(iu::ImageGpu_64f_C1& dst, double value) {iuprivate::math::fill(dst,value);}
void fill(iu::ImageGpu_64f_C2& dst, double2 value) {iuprivate::math::fill(dst,value);}
void fill(iu::ImageGpu_64f_C4& dst, double4 value) {iuprivate::math::fill(dst,value);}

void fill(iu::ImageCpu_64f_C1& dst, double value) {iuprivate::math::fill(dst,value);}
void fill(iu::ImageCpu_64f_C2& dst, double2 value) {iuprivate::math::fill(dst,value);}
void fill(iu::ImageCpu_64f_C4& dst, double4 value) {iuprivate::math::fill(dst,value);}

void fill(iu::LinearDeviceMemory_64f_C1& dst, double value) {iuprivate::math::fill(dst,value);}
void fill(iu::LinearDeviceMemory_64f_C2& dst, double2 value) {iuprivate::math::fill(dst,value);}
void fill(iu::LinearDeviceMemory_64f_C3& dst, double3 value) {iuprivate::math::fill(dst,value);}

void fill(iu::LinearHostMemory_64f_C1& dst, double value) {iuprivate::math::fill(dst,value);}
void fill(iu::LinearHostMemory_64f_C2& dst, double2 value) {iuprivate::math::fill(dst,value);}
void fill(iu::LinearHostMemory_64f_C3& dst, double3 value) {iuprivate::math::fill(dst,value);}

void fill(iu::VolumeGpu_64f_C1& dst, double value) {iuprivate::math::fill(dst,value);}
void fill(iu::VolumeGpu_64f_C2& dst, double2 value) {iuprivate::math::fill(dst,value);}
void fill(iu::VolumeGpu_64f_C3& dst, double3 value) {iuprivate::math::fill(dst,value);}

void fill(iu::VolumeCpu_64f_C1& dst, double value) {iuprivate::math::fill(dst,value);}
void fill(iu::VolumeCpu_64f_C2& dst, double2 value) {iuprivate::math::fill(dst,value);}
void fill(iu::VolumeCpu_64f_C3& dst, double3 value) {iuprivate::math::fill(dst,value);}

// min-max
void minMax(iu::ImageGpu_32f_C1& src, float& minVal, float& maxVal) {iuprivate::math::minMax(src,minVal,maxVal);}
void minMax(iu::VolumeGpu_32f_C1& src, float& minVal, float& maxVal) {iuprivate::math::minMax(src,minVal,maxVal);}

void minMax(iu::ImageCpu_32f_C1& src, float& minVal, float& maxVal) {iuprivate::math::minMax(src,minVal,maxVal);}
void minMax(iu::VolumeCpu_32f_C1& src, float& minVal, float& maxVal) {iuprivate::math::minMax(src,minVal,maxVal);}

void minMax(iu::LinearDeviceMemory_32f_C1& src, float& minVal, float& maxVal, unsigned int& minIdx, unsigned int& maxIdx) {iuprivate::math::minMax(src,minVal,maxVal,minIdx,maxIdx);}
void minMax(iu::LinearHostMemory_32f_C1& src, float& minVal, float& maxVal, unsigned int& minIdx, unsigned int& maxIdx) {iuprivate::math::minMax(src,minVal,maxVal,minIdx,maxIdx);}

void minMax(iu::ImageGpu_64f_C1& src, double& minVal, double& maxVal) {iuprivate::math::minMax(src,minVal,maxVal);}
void minMax(iu::VolumeGpu_64f_C1& src, double& minVal, double& maxVal) {iuprivate::math::minMax(src,minVal,maxVal);}

void minMax(iu::ImageCpu_64f_C1& src, double& minVal, double& maxVal) {iuprivate::math::minMax(src,minVal,maxVal);}
void minMax(iu::VolumeCpu_64f_C1& src, double& minVal, double& maxVal) {iuprivate::math::minMax(src,minVal,maxVal);}

void minMax(iu::LinearDeviceMemory_64f_C1& src, double& minVal, double& maxVal, unsigned int& minIdx, unsigned int& maxIdx) {iuprivate::math::minMax(src,minVal,maxVal,minIdx,maxIdx);}
void minMax(iu::LinearHostMemory_64f_C1& src, double& minVal, double& maxVal, unsigned int& minIdx, unsigned int& maxIdx) {iuprivate::math::minMax(src,minVal,maxVal,minIdx,maxIdx);}

//sum
void summation(iu::ImageGpu_32f_C1& src, float& sum) {iuprivate::math::summation(src,0.f,sum);}
void summation(iu::VolumeGpu_32f_C1& src, float& sum) {iuprivate::math::summation(src,0.f,sum);}
void summation(iu::LinearDeviceMemory_32f_C1& src, float& sum) {iuprivate::math::summation(src,0.f,sum);}

void summation(iu::ImageCpu_32f_C1& src, float& sum) {iuprivate::math::summation(src,0.f,sum);}
void summation(iu::VolumeCpu_32f_C1& src, float& sum) {iuprivate::math::summation(src,0.f,sum);}
void summation(iu::LinearHostMemory_32f_C1& src, float& sum) {iuprivate::math::summation(src,0.f,sum);}

void summation(iu::ImageGpu_64f_C1& src, double& sum) {iuprivate::math::summation(src,0.0,sum);}
void summation(iu::VolumeGpu_64f_C1& src, double& sum) {iuprivate::math::summation(src,0.0,sum);}
void summation(iu::LinearDeviceMemory_64f_C1& src, double& sum) {iuprivate::math::summation(src,0.0,sum);}

void summation(iu::ImageCpu_64f_C1& src, double& sum) {iuprivate::math::summation(src,0.0,sum);}
void summation(iu::VolumeCpu_64f_C1& src, double& sum) {iuprivate::math::summation(src,0.0,sum);}
void summation(iu::LinearHostMemory_64f_C1& src, double& sum) {iuprivate::math::summation(src,0.0,sum);}


// L1-norm
void normDiffL1(iu::ImageGpu_32f_C1& src, iu::ImageGpu_32f_C1& ref, float& norm) {iuprivate::math::normDiffL1(src,ref,norm);}
void normDiffL1(iu::ImageGpu_32f_C1& src, float& ref, float& norm) {iuprivate::math::normDiffL1(src,ref,norm);}
void normDiffL1(iu::VolumeGpu_32f_C1& src, iu::VolumeGpu_32f_C1& ref, float& norm) {iuprivate::math::normDiffL1(src,ref,norm);}
void normDiffL1(iu::VolumeGpu_32f_C1& src, float& ref, float& norm) {iuprivate::math::normDiffL1(src,ref,norm);}

void normDiffL1(iu::ImageCpu_32f_C1& src, iu::ImageCpu_32f_C1& ref, float& norm) {iuprivate::math::normDiffL1(src,ref,norm);}
void normDiffL1(iu::ImageCpu_32f_C1& src, float& ref, float& norm) {iuprivate::math::normDiffL1(src,ref,norm);}
void normDiffL1(iu::VolumeCpu_32f_C1& src, iu::VolumeCpu_32f_C1& ref, float& norm) {iuprivate::math::normDiffL1(src,ref,norm);}
void normDiffL1(iu::VolumeCpu_32f_C1& src, float& ref, float& norm) {iuprivate::math::normDiffL1(src,ref,norm);}

void normDiffL1(iu::ImageGpu_64f_C1& src, iu::ImageGpu_64f_C1& ref, double& norm) {iuprivate::math::normDiffL1(src,ref,norm);}
void normDiffL1(iu::ImageGpu_64f_C1& src, double& ref, double& norm) {iuprivate::math::normDiffL1(src,ref,norm);}
void normDiffL1(iu::VolumeGpu_64f_C1& src, iu::VolumeGpu_64f_C1& ref, double& norm) {iuprivate::math::normDiffL1(src,ref,norm);}
void normDiffL1(iu::VolumeGpu_64f_C1& src, double& ref, double& norm) {iuprivate::math::normDiffL1(src,ref,norm);}

void normDiffL1(iu::ImageCpu_64f_C1& src, iu::ImageCpu_64f_C1& ref, double& norm) {iuprivate::math::normDiffL1(src,ref,norm);}
void normDiffL1(iu::ImageCpu_64f_C1& src, double& ref, double& norm) {iuprivate::math::normDiffL1(src,ref,norm);}
void normDiffL1(iu::VolumeCpu_64f_C1& src, iu::VolumeCpu_64f_C1& ref, double& norm) {iuprivate::math::normDiffL1(src,ref,norm);}
void normDiffL1(iu::VolumeCpu_64f_C1& src, double& ref, double& norm) {iuprivate::math::normDiffL1(src,ref,norm);}


// L2-norm
void normDiffL2(iu::ImageGpu_32f_C1& src, iu::ImageGpu_32f_C1& ref, float& norm) {iuprivate::math::normDiffL2(src,ref,norm);}
void normDiffL2(iu::ImageGpu_32f_C1& src, float& ref, float& norm) {iuprivate::math::normDiffL2(src,ref,norm);}
void normDiffL2(iu::VolumeGpu_32f_C1& src, iu::VolumeGpu_32f_C1& ref, float& norm) {iuprivate::math::normDiffL2(src,ref,norm);}
void normDiffL2(iu::VolumeGpu_32f_C1& src, float& ref, float& norm) {iuprivate::math::normDiffL2(src,ref,norm);}

void normDiffL2(iu::ImageCpu_32f_C1& src, iu::ImageCpu_32f_C1& ref, float& norm) {iuprivate::math::normDiffL2(src,ref,norm);}
void normDiffL2(iu::ImageCpu_32f_C1& src, float& ref, float& norm) {iuprivate::math::normDiffL2(src,ref,norm);}
void normDiffL2(iu::VolumeCpu_32f_C1& src, iu::VolumeCpu_32f_C1& ref, float& norm) {iuprivate::math::normDiffL2(src,ref,norm);}
void normDiffL2(iu::VolumeCpu_32f_C1& src, float& ref, float& norm) {iuprivate::math::normDiffL2(src,ref,norm);}

void normDiffL2(iu::ImageGpu_64f_C1& src, iu::ImageGpu_64f_C1& ref, double& norm) {iuprivate::math::normDiffL2(src,ref,norm);}
void normDiffL2(iu::ImageGpu_64f_C1& src, double& ref, double& norm) {iuprivate::math::normDiffL2(src,ref,norm);}
void normDiffL2(iu::VolumeGpu_64f_C1& src, iu::VolumeGpu_64f_C1& ref, double& norm) {iuprivate::math::normDiffL2(src,ref,norm);}
void normDiffL2(iu::VolumeGpu_64f_C1& src, double& ref, double& norm) {iuprivate::math::normDiffL2(src,ref,norm);}

void normDiffL2(iu::ImageCpu_64f_C1& src, iu::ImageCpu_64f_C1& ref, double& norm) {iuprivate::math::normDiffL2(src,ref,norm);}
void normDiffL2(iu::ImageCpu_64f_C1& src, double& ref, double& norm) {iuprivate::math::normDiffL2(src,ref,norm);}
void normDiffL2(iu::VolumeCpu_64f_C1& src, iu::VolumeCpu_64f_C1& ref, double& norm) {iuprivate::math::normDiffL2(src,ref,norm);}
void normDiffL2(iu::VolumeCpu_64f_C1& src, double& ref, double& norm) {iuprivate::math::normDiffL2(src,ref,norm);}

// MSE
void mse(iu::ImageGpu_32f_C1& src, iu::ImageGpu_32f_C1& ref, float& mse) {iuprivate::math::mse(src,ref,mse);}
void mse(iu::VolumeGpu_32f_C1& src, iu::VolumeGpu_32f_C1& ref, float& mse) {iuprivate::math::mse(src,ref,mse);}

void mse(iu::ImageCpu_32f_C1& src, iu::ImageCpu_32f_C1& ref, float& mse) {iuprivate::math::mse(src,ref,mse);}
void mse(iu::VolumeCpu_32f_C1& src, iu::VolumeCpu_32f_C1& ref, float& mse) {iuprivate::math::mse(src,ref,mse);}

void mse(iu::ImageGpu_64f_C1& src, iu::ImageGpu_64f_C1& ref, double& mse) {iuprivate::math::mse(src,ref,mse);}
void mse(iu::VolumeGpu_64f_C1& src, iu::VolumeGpu_64f_C1& ref, double& mse) {iuprivate::math::mse(src,ref,mse);}

void mse(iu::ImageCpu_64f_C1& src, iu::ImageCpu_64f_C1& ref, double& mse) {iuprivate::math::mse(src,ref,mse);}
void mse(iu::VolumeCpu_64f_C1& src, iu::VolumeCpu_64f_C1& ref, double& mse) {iuprivate::math::mse(src,ref,mse);}

// split planes
void splitPlanes(iu::VolumeCpu_32f_C2& src, iu::VolumeCpu_32f_C1& dst1, iu::VolumeCpu_32f_C1& dst2) {iuprivate::math::splitPlanes(src, dst1, dst2);}
void splitPlanes(iu::VolumeGpu_32f_C2& src, iu::VolumeGpu_32f_C1& dst1, iu::VolumeGpu_32f_C1& dst2) {iuprivate::math::splitPlanes(src, dst1, dst2);}
void splitPlanes(iu::ImageCpu_32f_C2& src, iu::ImageCpu_32f_C1& dst1, iu::ImageCpu_32f_C1& dst2) {iuprivate::math::splitPlanes(src, dst1, dst2);}
void splitPlanes(iu::ImageGpu_32f_C2& src, iu::ImageGpu_32f_C1& dst1, iu::ImageGpu_32f_C1& dst2) {iuprivate::math::splitPlanes(src, dst1, dst2);}
void splitPlanes(iu::LinearHostMemory_32f_C2& src, iu::LinearHostMemory_32f_C1& dst1, iu::LinearHostMemory_32f_C1& dst2) {iuprivate::math::splitPlanes(src, dst1, dst2);}
void splitPlanes(iu::LinearDeviceMemory_32f_C2& src, iu::LinearDeviceMemory_32f_C1& dst1, iu::LinearDeviceMemory_32f_C1& dst2) {iuprivate::math::splitPlanes(src, dst1, dst2);}

void splitPlanes(iu::VolumeCpu_32f_C3& src, iu::VolumeCpu_32f_C1& dst1, iu::VolumeCpu_32f_C1& dst2, iu::VolumeCpu_32f_C1& dst3){iuprivate::math::splitPlanes(src, dst1, dst2, dst3);}
void splitPlanes(iu::VolumeGpu_32f_C3& src, iu::VolumeGpu_32f_C1& dst1, iu::VolumeGpu_32f_C1& dst2, iu::VolumeGpu_32f_C1& dst3){iuprivate::math::splitPlanes(src, dst1, dst2, dst3);}
void splitPlanes(iu::ImageCpu_32f_C3& src, iu::ImageCpu_32f_C1& dst1, iu::ImageCpu_32f_C1& dst2, iu::ImageCpu_32f_C1& dst3){iuprivate::math::splitPlanes(src, dst1, dst2, dst3);}
void splitPlanes(iu::ImageGpu_32f_C3& src, iu::ImageGpu_32f_C1& dst1, iu::ImageGpu_32f_C1& dst2, iu::ImageGpu_32f_C1& dst3){iuprivate::math::splitPlanes(src, dst1, dst2, dst3);}
void splitPlanes(iu::LinearHostMemory_32f_C3& src, iu::LinearHostMemory_32f_C1& dst1, iu::LinearHostMemory_32f_C1& dst2, iu::LinearHostMemory_32f_C1& dst3){iuprivate::math::splitPlanes(src, dst1, dst2, dst3);}
void splitPlanes(iu::LinearDeviceMemory_32f_C3& src, iu::LinearDeviceMemory_32f_C1& dst1, iu::LinearDeviceMemory_32f_C1& dst2, iu::LinearDeviceMemory_32f_C1& dst3){iuprivate::math::splitPlanes(src, dst1, dst2, dst3);}

void splitPlanes(iu::VolumeCpu_64f_C2& src, iu::VolumeCpu_64f_C1& dst1, iu::VolumeCpu_64f_C1& dst2) {iuprivate::math::splitPlanes(src, dst1, dst2);}
void splitPlanes(iu::VolumeGpu_64f_C2& src, iu::VolumeGpu_64f_C1& dst1, iu::VolumeGpu_64f_C1& dst2) {iuprivate::math::splitPlanes(src, dst1, dst2);}
void splitPlanes(iu::ImageCpu_64f_C2& src, iu::ImageCpu_64f_C1& dst1, iu::ImageCpu_64f_C1& dst2) {iuprivate::math::splitPlanes(src, dst1, dst2);}
void splitPlanes(iu::ImageGpu_64f_C2& src, iu::ImageGpu_64f_C1& dst1, iu::ImageGpu_64f_C1& dst2) {iuprivate::math::splitPlanes(src, dst1, dst2);}
void splitPlanes(iu::LinearHostMemory_64f_C2& src, iu::LinearHostMemory_64f_C1& dst1, iu::LinearHostMemory_64f_C1& dst2) {iuprivate::math::splitPlanes(src, dst1, dst2);}
void splitPlanes(iu::LinearDeviceMemory_64f_C2& src, iu::LinearDeviceMemory_64f_C1& dst1, iu::LinearDeviceMemory_64f_C1& dst2) {iuprivate::math::splitPlanes(src, dst1, dst2);}

void splitPlanes(iu::VolumeCpu_64f_C3& src, iu::VolumeCpu_64f_C1& dst1, iu::VolumeCpu_64f_C1& dst2, iu::VolumeCpu_64f_C1& dst3){iuprivate::math::splitPlanes(src, dst1, dst2, dst3);}
void splitPlanes(iu::VolumeGpu_64f_C3& src, iu::VolumeGpu_64f_C1& dst1, iu::VolumeGpu_64f_C1& dst2, iu::VolumeGpu_64f_C1& dst3){iuprivate::math::splitPlanes(src, dst1, dst2, dst3);}
void splitPlanes(iu::ImageCpu_64f_C3& src, iu::ImageCpu_64f_C1& dst1, iu::ImageCpu_64f_C1& dst2, iu::ImageCpu_64f_C1& dst3){iuprivate::math::splitPlanes(src, dst1, dst2, dst3);}
void splitPlanes(iu::ImageGpu_64f_C3& src, iu::ImageGpu_64f_C1& dst1, iu::ImageGpu_64f_C1& dst2, iu::ImageGpu_64f_C1& dst3){iuprivate::math::splitPlanes(src, dst1, dst2, dst3);}
void splitPlanes(iu::LinearHostMemory_64f_C3& src, iu::LinearHostMemory_64f_C1& dst1, iu::LinearHostMemory_64f_C1& dst2, iu::LinearHostMemory_64f_C1& dst3){iuprivate::math::splitPlanes(src, dst1, dst2, dst3);}
void splitPlanes(iu::LinearDeviceMemory_64f_C3& src, iu::LinearDeviceMemory_64f_C1& dst1, iu::LinearDeviceMemory_64f_C1& dst2, iu::LinearDeviceMemory_64f_C1& dst3){iuprivate::math::splitPlanes(src, dst1, dst2, dst3);}

// combine planes
void combinePlanes(iu::VolumeCpu_32f_C1& src1, iu::VolumeCpu_32f_C1& src2, iu::VolumeCpu_32f_C2& dst) {iuprivate::math::combinePlanes(src1, src2, dst);}
void combinePlanes(iu::VolumeGpu_32f_C1& src1, iu::VolumeGpu_32f_C1& src2, iu::VolumeGpu_32f_C2& dst) {iuprivate::math::combinePlanes(src1, src2, dst);}
void combinePlanes(iu::ImageCpu_32f_C1& src1, iu::ImageCpu_32f_C1& src2, iu::ImageCpu_32f_C2& dst) {iuprivate::math::combinePlanes(src1, src2, dst);}
void combinePlanes(iu::ImageGpu_32f_C1& src1, iu::ImageGpu_32f_C1& src2, iu::ImageGpu_32f_C2& dst) {iuprivate::math::combinePlanes(src1, src2, dst);}
void combinePlanes(iu::LinearHostMemory_32f_C1& src1, iu::LinearHostMemory_32f_C1& src2, iu::LinearHostMemory_32f_C2& dst) {iuprivate::math::combinePlanes(src1, src2, dst);}
void combinePlanes(iu::LinearDeviceMemory_32f_C1& src1, iu::LinearDeviceMemory_32f_C1& src2, iu::LinearDeviceMemory_32f_C2& dst) {iuprivate::math::combinePlanes(src1, src2, dst);}

void combinePlanes(iu::VolumeCpu_32f_C1& src1, iu::VolumeCpu_32f_C1& src2, iu::VolumeCpu_32f_C1& src3, iu::VolumeCpu_32f_C3& dst){iuprivate::math::combinePlanes(src1, src2, src3, dst);}
void combinePlanes(iu::VolumeGpu_32f_C1& src1, iu::VolumeGpu_32f_C1& src2, iu::VolumeGpu_32f_C1& src3, iu::VolumeGpu_32f_C3& dst){iuprivate::math::combinePlanes(src1, src2, src3, dst);}
void combinePlanes(iu::ImageCpu_32f_C1& src1, iu::ImageCpu_32f_C1& src2, iu::ImageCpu_32f_C1& src3, iu::ImageCpu_32f_C3& dst){iuprivate::math::combinePlanes(src1, src2, src3, dst);}
void combinePlanes(iu::ImageGpu_32f_C1& src1, iu::ImageGpu_32f_C1& src2, iu::ImageGpu_32f_C1& src3, iu::ImageGpu_32f_C3& dst){iuprivate::math::combinePlanes(src1, src2, src3, dst);}
void combinePlanes(iu::LinearHostMemory_32f_C1& src1, iu::LinearHostMemory_32f_C1& src2, iu::LinearHostMemory_32f_C1& src3, iu::LinearHostMemory_32f_C3& dst){iuprivate::math::combinePlanes(src1, src2, src3, dst);}
void combinePlanes(iu::LinearDeviceMemory_32f_C1& src1, iu::LinearDeviceMemory_32f_C1& src2, iu::LinearDeviceMemory_32f_C1& src3, iu::LinearDeviceMemory_32f_C3& dst){iuprivate::math::combinePlanes(src1, src2, src3, dst);}

void combinePlanes(iu::VolumeCpu_64f_C1& src1, iu::VolumeCpu_64f_C1& src2, iu::VolumeCpu_64f_C2& dst) {iuprivate::math::combinePlanes(src1, src2, dst);}
void combinePlanes(iu::VolumeGpu_64f_C1& src1, iu::VolumeGpu_64f_C1& src2, iu::VolumeGpu_64f_C2& dst) {iuprivate::math::combinePlanes(src1, src2, dst);}
void combinePlanes(iu::ImageCpu_64f_C1& src1, iu::ImageCpu_64f_C1& src2, iu::ImageCpu_64f_C2& dst) {iuprivate::math::combinePlanes(src1, src2, dst);}
void combinePlanes(iu::ImageGpu_64f_C1& src1, iu::ImageGpu_64f_C1& src2, iu::ImageGpu_64f_C2& dst) {iuprivate::math::combinePlanes(src1, src2, dst);}
void combinePlanes(iu::LinearHostMemory_64f_C1& src1, iu::LinearHostMemory_64f_C1& src2, iu::LinearHostMemory_64f_C2& dst) {iuprivate::math::combinePlanes(src1, src2, dst);}
void combinePlanes(iu::LinearDeviceMemory_64f_C1& src1, iu::LinearDeviceMemory_64f_C1& src2, iu::LinearDeviceMemory_64f_C2& dst) {iuprivate::math::combinePlanes(src1, src2, dst);}

void combinePlanes(iu::VolumeCpu_64f_C1& src1, iu::VolumeCpu_64f_C1& src2, iu::VolumeCpu_64f_C1& src3, iu::VolumeCpu_64f_C3& dst){iuprivate::math::combinePlanes(src1, src2, src3, dst);}
void combinePlanes(iu::VolumeGpu_64f_C1& src1, iu::VolumeGpu_64f_C1& src2, iu::VolumeGpu_64f_C1& src3, iu::VolumeGpu_64f_C3& dst){iuprivate::math::combinePlanes(src1, src2, src3, dst);}
void combinePlanes(iu::ImageCpu_64f_C1& src1, iu::ImageCpu_64f_C1& src2, iu::ImageCpu_64f_C1& src3, iu::ImageCpu_64f_C3& dst){iuprivate::math::combinePlanes(src1, src2, src3, dst);}
void combinePlanes(iu::ImageGpu_64f_C1& src1, iu::ImageGpu_64f_C1& src2, iu::ImageGpu_64f_C1& src3, iu::ImageGpu_64f_C3& dst){iuprivate::math::combinePlanes(src1, src2, src3, dst);}
void combinePlanes(iu::LinearHostMemory_64f_C1& src1, iu::LinearHostMemory_64f_C1& src2, iu::LinearHostMemory_64f_C1& src3, iu::LinearHostMemory_64f_C3& dst){iuprivate::math::combinePlanes(src1, src2, src3, dst);}
void combinePlanes(iu::LinearDeviceMemory_64f_C1& src1, iu::LinearDeviceMemory_64f_C1& src2, iu::LinearDeviceMemory_64f_C1& src3, iu::LinearDeviceMemory_64f_C3& dst){iuprivate::math::combinePlanes(src1, src2, src3, dst);}

namespace complex {
// abs
void abs(iu::VolumeCpu_32f_C2& complex, iu::VolumeCpu_32f_C1& real) {iuprivate::math::complex::abs(complex, real);}
void abs(iu::VolumeGpu_32f_C2& complex, iu::VolumeGpu_32f_C1& real) {iuprivate::math::complex::abs(complex, real);}
void abs(iu::ImageCpu_32f_C2& complex, iu::ImageCpu_32f_C1& real) {iuprivate::math::complex::abs(complex, real);}
void abs(iu::ImageGpu_32f_C2& complex, iu::ImageGpu_32f_C1& real) {iuprivate::math::complex::abs(complex, real);}
void abs(iu::LinearHostMemory_32f_C2& complex, iu::LinearHostMemory_32f_C1& real) {iuprivate::math::complex::abs(complex, real);}
void abs(iu::LinearDeviceMemory_32f_C2& complex, iu::LinearDeviceMemory_32f_C1& real) {iuprivate::math::complex::abs(complex, real);}

void abs(iu::VolumeCpu_64f_C2& complex, iu::VolumeCpu_64f_C1& real) {iuprivate::math::complex::abs(complex, real);}
void abs(iu::VolumeGpu_64f_C2& complex, iu::VolumeGpu_64f_C1& real) {iuprivate::math::complex::abs(complex, real);}
void abs(iu::ImageCpu_64f_C2& complex, iu::ImageCpu_64f_C1& real) {iuprivate::math::complex::abs(complex, real);}
void abs(iu::ImageGpu_64f_C2& complex, iu::ImageGpu_64f_C1& real) {iuprivate::math::complex::abs(complex, real);}
void abs(iu::LinearHostMemory_64f_C2& complex, iu::LinearHostMemory_64f_C1& real) {iuprivate::math::complex::abs(complex, real);}
void abs(iu::LinearDeviceMemory_64f_C2& complex, iu::LinearDeviceMemory_64f_C1& real) {iuprivate::math::complex::abs(complex, real);}

// real
void real(iu::VolumeCpu_32f_C2& complex, iu::VolumeCpu_32f_C1& real) {iuprivate::math::complex::real(complex, real);}
void real(iu::VolumeGpu_32f_C2& complex, iu::VolumeGpu_32f_C1& real) {iuprivate::math::complex::real(complex, real);}
void real(iu::ImageCpu_32f_C2& complex, iu::ImageCpu_32f_C1& real) {iuprivate::math::complex::real(complex, real);}
void real(iu::ImageGpu_32f_C2& complex, iu::ImageGpu_32f_C1& real) {iuprivate::math::complex::real(complex, real);}
void real(iu::LinearHostMemory_32f_C2& complex, iu::LinearHostMemory_32f_C1& real) {iuprivate::math::complex::real(complex, real);}
void real(iu::LinearDeviceMemory_32f_C2& complex, iu::LinearDeviceMemory_32f_C1& real) {iuprivate::math::complex::real(complex, real);}

void real(iu::VolumeCpu_64f_C2& complex, iu::VolumeCpu_64f_C1& real) {iuprivate::math::complex::real(complex, real);}
void real(iu::VolumeGpu_64f_C2& complex, iu::VolumeGpu_64f_C1& real) {iuprivate::math::complex::real(complex, real);}
void real(iu::ImageCpu_64f_C2& complex, iu::ImageCpu_64f_C1& real) {iuprivate::math::complex::real(complex, real);}
void real(iu::ImageGpu_64f_C2& complex, iu::ImageGpu_64f_C1& real) {iuprivate::math::complex::real(complex, real);}
void real(iu::LinearHostMemory_64f_C2& complex, iu::LinearHostMemory_64f_C1& real) {iuprivate::math::complex::real(complex, real);}
void real(iu::LinearDeviceMemory_64f_C2& complex, iu::LinearDeviceMemory_64f_C1& real) {iuprivate::math::complex::real(complex, real);}

// imag
void imag(iu::VolumeCpu_32f_C2& complex, iu::VolumeCpu_32f_C1& real) {iuprivate::math::complex::imag(complex, real);}
void imag(iu::VolumeGpu_32f_C2& complex, iu::VolumeGpu_32f_C1& real) {iuprivate::math::complex::imag(complex, real);}
void imag(iu::ImageCpu_32f_C2& complex, iu::ImageCpu_32f_C1& real) {iuprivate::math::complex::imag(complex, real);}
void imag(iu::ImageGpu_32f_C2& complex, iu::ImageGpu_32f_C1& real) {iuprivate::math::complex::imag(complex, real);}
void imag(iu::LinearHostMemory_32f_C2& complex, iu::LinearHostMemory_32f_C1& real) {iuprivate::math::complex::imag(complex, real);}
void imag(iu::LinearDeviceMemory_32f_C2& complex, iu::LinearDeviceMemory_32f_C1& real) {iuprivate::math::complex::imag(complex, real);}

void imag(iu::VolumeCpu_64f_C2& complex, iu::VolumeCpu_64f_C1& real) {iuprivate::math::complex::imag(complex, real);}
void imag(iu::VolumeGpu_64f_C2& complex, iu::VolumeGpu_64f_C1& real) {iuprivate::math::complex::imag(complex, real);}
void imag(iu::ImageCpu_64f_C2& complex, iu::ImageCpu_64f_C1& real) {iuprivate::math::complex::imag(complex, real);}
void imag(iu::ImageGpu_64f_C2& complex, iu::ImageGpu_64f_C1& real) {iuprivate::math::complex::imag(complex, real);}
void imag(iu::LinearHostMemory_64f_C2& complex, iu::LinearHostMemory_64f_C1& real) {iuprivate::math::complex::imag(complex, real);}
void imag(iu::LinearDeviceMemory_64f_C2& complex, iu::LinearDeviceMemory_64f_C1& real) {iuprivate::math::complex::imag(complex, real);}

// phase
void phase(iu::VolumeCpu_32f_C2& complex, iu::VolumeCpu_32f_C1& real) {iuprivate::math::complex::phase(complex, real);}
void phase(iu::VolumeGpu_32f_C2& complex, iu::VolumeGpu_32f_C1& real) {iuprivate::math::complex::phase(complex, real);}
void phase(iu::ImageCpu_32f_C2& complex, iu::ImageCpu_32f_C1& real) {iuprivate::math::complex::phase(complex, real);}
void phase(iu::ImageGpu_32f_C2& complex, iu::ImageGpu_32f_C1& real) {iuprivate::math::complex::phase(complex, real);}
void phase(iu::LinearHostMemory_32f_C2& complex, iu::LinearHostMemory_32f_C1& real) {iuprivate::math::complex::phase(complex, real);}
void phase(iu::LinearDeviceMemory_32f_C2& complex, iu::LinearDeviceMemory_32f_C1& real) {iuprivate::math::complex::phase(complex, real);}

void phase(iu::VolumeCpu_64f_C2& complex, iu::VolumeCpu_64f_C1& real) {iuprivate::math::complex::phase(complex, real);}
void phase(iu::VolumeGpu_64f_C2& complex, iu::VolumeGpu_64f_C1& real) {iuprivate::math::complex::phase(complex, real);}
void phase(iu::ImageCpu_64f_C2& complex, iu::ImageCpu_64f_C1& real) {iuprivate::math::complex::phase(complex, real);}
void phase(iu::ImageGpu_64f_C2& complex, iu::ImageGpu_64f_C1& real) {iuprivate::math::complex::phase(complex, real);}
void phase(iu::LinearHostMemory_64f_C2& complex, iu::LinearHostMemory_64f_C1& real) {iuprivate::math::complex::phase(complex, real);}
void phase(iu::LinearDeviceMemory_64f_C2& complex, iu::LinearDeviceMemory_64f_C1& real) {iuprivate::math::complex::phase(complex, real);}

// scale
void scale(iu::VolumeCpu_32f_C2& complex_src, const float& scale, iu::VolumeCpu_32f_C2& complex_dst) {iuprivate::math::mulC(complex_src, make_float2(scale, scale), complex_dst);}
void scale(iu::VolumeGpu_32f_C2& complex_src, const float& scale, iu::VolumeGpu_32f_C2& complex_dst) {iuprivate::math::mulC(complex_src, make_float2(scale, scale), complex_dst);}
void scale(iu::ImageCpu_32f_C2& complex_src, const float& scale, iu::ImageCpu_32f_C2& complex_dst) {iuprivate::math::mulC(complex_src, make_float2(scale, scale), complex_dst);}
void scale(iu::ImageGpu_32f_C2& complex_src, const float& scale, iu::ImageGpu_32f_C2& complex_dst) {iuprivate::math::mulC(complex_src, make_float2(scale, scale), complex_dst);}
void scale(iu::LinearHostMemory_32f_C2& complex_src, const float& scale, iu::LinearHostMemory_32f_C2& complex_dst) {iuprivate::math::mulC(complex_src, make_float2(scale, scale), complex_dst);}
void scale(iu::LinearDeviceMemory_32f_C2& complex_src, const float& scale, iu::LinearDeviceMemory_32f_C2& complex_dst) {iuprivate::math::mulC(complex_src, make_float2(scale, scale), complex_dst);}

void scale(iu::VolumeCpu_64f_C2& complex_src, const double& scale, iu::VolumeCpu_64f_C2& complex_dst) {iuprivate::math::mulC(complex_src, make_double2(scale, scale), complex_dst);}
void scale(iu::VolumeGpu_64f_C2& complex_src, const double& scale, iu::VolumeGpu_64f_C2& complex_dst) {iuprivate::math::mulC(complex_src, make_double2(scale, scale), complex_dst);}
void scale(iu::ImageCpu_64f_C2& complex_src, const double& scale, iu::ImageCpu_64f_C2& complex_dst) {iuprivate::math::mulC(complex_src, make_double2(scale, scale), complex_dst);}
void scale(iu::ImageGpu_64f_C2& complex_src, const double& scale, iu::ImageGpu_64f_C2& complex_dst) {iuprivate::math::mulC(complex_src, make_double2(scale, scale), complex_dst);}
void scale(iu::LinearHostMemory_64f_C2& complex_src, const double& scale, iu::LinearHostMemory_64f_C2& complex_dst) {iuprivate::math::mulC(complex_src, make_double2(scale, scale), complex_dst);}
void scale(iu::LinearDeviceMemory_64f_C2& complex_src, const double& scale, iu::LinearDeviceMemory_64f_C2& complex_dst) {iuprivate::math::mulC(complex_src, make_double2(scale, scale), complex_dst);}

// multiply complex with real
void multiply(iu::VolumeCpu_32f_C2& complex_src, iu::VolumeCpu_32f_C1& real, iu::VolumeCpu_32f_C2& complex_dst)
{
  iuprivate::math::combinePlanes(real, real, complex_dst);
  iuprivate::math::mul(complex_src, complex_dst, complex_dst);
}
void multiply(iu::VolumeGpu_32f_C2& complex_src, iu::VolumeGpu_32f_C1& real, iu::VolumeGpu_32f_C2& complex_dst)
{
  iuprivate::math::combinePlanes(real, real, complex_dst);
  iuprivate::math::mul(complex_src, complex_dst, complex_dst);
}
void multiply(iu::ImageCpu_32f_C2& complex_src, iu::ImageCpu_32f_C1& real, iu::ImageCpu_32f_C2& complex_dst)
{
  iuprivate::math::combinePlanes(real, real, complex_dst);
  iuprivate::math::mul(complex_src, complex_dst, complex_dst);
}
void multiply(iu::ImageGpu_32f_C2& complex_src, iu::ImageGpu_32f_C1& real, iu::ImageGpu_32f_C2& complex_dst)
{
  iuprivate::math::combinePlanes(real, real, complex_dst);
  iuprivate::math::mul(complex_src, complex_dst, complex_dst);
}

void multiply(iu::LinearHostMemory_32f_C2& complex_src, iu::LinearHostMemory_32f_C1& real, iu::LinearHostMemory_32f_C2& complex_dst)
{
  iuprivate::math::combinePlanes(real, real, complex_dst);
  iuprivate::math::mul(complex_src, complex_dst, complex_dst);
}
void multiply(iu::LinearDeviceMemory_32f_C2& complex_src, iu::LinearDeviceMemory_32f_C1& real, iu::LinearDeviceMemory_32f_C2& complex_dst)
{
  iuprivate::math::combinePlanes(real, real, complex_dst);
  iuprivate::math::mul(complex_src, complex_dst, complex_dst);
}

void multiply(iu::VolumeCpu_64f_C2& complex_src, iu::VolumeCpu_64f_C1& real, iu::VolumeCpu_64f_C2& complex_dst)
{
  iuprivate::math::combinePlanes(real, real, complex_dst);
  iuprivate::math::mul(complex_src, complex_dst, complex_dst);
}
void multiply(iu::VolumeGpu_64f_C2& complex_src, iu::VolumeGpu_64f_C1& real, iu::VolumeGpu_64f_C2& complex_dst)
{
  iuprivate::math::combinePlanes(real, real, complex_dst);
  iuprivate::math::mul(complex_src, complex_dst, complex_dst);
}
void multiply(iu::ImageCpu_64f_C2& complex_src, iu::ImageCpu_64f_C1& real, iu::ImageCpu_64f_C2& complex_dst)
{
  iuprivate::math::combinePlanes(real, real, complex_dst);
  iuprivate::math::mul(complex_src, complex_dst, complex_dst);
}
void multiply(iu::ImageGpu_64f_C2& complex_src, iu::ImageGpu_64f_C1& real, iu::ImageGpu_64f_C2& complex_dst)
{
  iuprivate::math::combinePlanes(real, real, complex_dst);
  iuprivate::math::mul(complex_src, complex_dst, complex_dst);
}

void multiply(iu::LinearHostMemory_64f_C2& complex_src, iu::LinearHostMemory_64f_C1& real, iu::LinearHostMemory_64f_C2& complex_dst)
{
  iuprivate::math::combinePlanes(real, real, complex_dst);
  iuprivate::math::mul(complex_src, complex_dst, complex_dst);
}
void multiply(iu::LinearDeviceMemory_64f_C2& complex_src, iu::LinearDeviceMemory_64f_C1& real, iu::LinearDeviceMemory_64f_C2& complex_dst)
{
  iuprivate::math::combinePlanes(real, real, complex_dst);
  iuprivate::math::mul(complex_src, complex_dst, complex_dst);
}


// multiply complex with complex
void multiply(iu::VolumeCpu_32f_C2& complex_src1, iu::VolumeCpu_32f_C2& complex_src2, iu::VolumeCpu_32f_C2& complex_dst){iuprivate::math::complex::multiply(complex_src1,complex_src2,complex_dst);}
void multiply(iu::VolumeGpu_32f_C2& complex_src1, iu::VolumeGpu_32f_C2& complex_src2, iu::VolumeGpu_32f_C2& complex_dst){iuprivate::math::complex::multiply(complex_src1,complex_src2,complex_dst);}
void multiply(iu::ImageCpu_32f_C2& complex_src1, iu::ImageCpu_32f_C2& complex_src2, iu::ImageCpu_32f_C2& complex_dst){iuprivate::math::complex::multiply(complex_src1,complex_src2,complex_dst);}
void multiply(iu::ImageGpu_32f_C2& complex_src1, iu::ImageGpu_32f_C2& complex_src2, iu::ImageGpu_32f_C2& complex_dst){iuprivate::math::complex::multiply(complex_src1,complex_src2,complex_dst);}
void multiply(iu::LinearHostMemory_32f_C2& complex_src1, iu::LinearHostMemory_32f_C2& complex_src2, iu::LinearHostMemory_32f_C2& complex_dst){iuprivate::math::complex::multiply(complex_src1,complex_src2,complex_dst);}
void multiply(iu::LinearDeviceMemory_32f_C2& complex_src1, iu::LinearDeviceMemory_32f_C2& complex_src2, iu::LinearDeviceMemory_32f_C2& complex_dst){iuprivate::math::complex::multiply(complex_src1,complex_src2,complex_dst);}

void multiply(iu::VolumeCpu_64f_C2& complex_src1, iu::VolumeCpu_64f_C2& complex_src2, iu::VolumeCpu_64f_C2& complex_dst){iuprivate::math::complex::multiply(complex_src1,complex_src2,complex_dst);}
void multiply(iu::VolumeGpu_64f_C2& complex_src1, iu::VolumeGpu_64f_C2& complex_src2, iu::VolumeGpu_64f_C2& complex_dst){iuprivate::math::complex::multiply(complex_src1,complex_src2,complex_dst);}
void multiply(iu::ImageCpu_64f_C2& complex_src1, iu::ImageCpu_64f_C2& complex_src2, iu::ImageCpu_64f_C2& complex_dst){iuprivate::math::complex::multiply(complex_src1,complex_src2,complex_dst);}
void multiply(iu::ImageGpu_64f_C2& complex_src1, iu::ImageGpu_64f_C2& complex_src2, iu::ImageGpu_64f_C2& complex_dst){iuprivate::math::complex::multiply(complex_src1,complex_src2,complex_dst);}
void multiply(iu::LinearHostMemory_64f_C2& complex_src1, iu::LinearHostMemory_64f_C2& complex_src2, iu::LinearHostMemory_64f_C2& complex_dst){iuprivate::math::complex::multiply(complex_src1,complex_src2,complex_dst);}
void multiply(iu::LinearDeviceMemory_64f_C2& complex_src1, iu::LinearDeviceMemory_64f_C2& complex_src2, iu::LinearDeviceMemory_64f_C2& complex_dst){iuprivate::math::complex::multiply(complex_src1,complex_src2,complex_dst);}

// multiply complex with complex conjugate
void multiplyConjugate(iu::VolumeCpu_32f_C2& complex_src1, iu::VolumeCpu_32f_C2& complex_src2, iu::VolumeCpu_32f_C2& complex_dst){iuprivate::math::complex::multiplyConjugate(complex_src1,complex_src2,complex_dst);}
void multiplyConjugate(iu::VolumeGpu_32f_C2& complex_src1, iu::VolumeGpu_32f_C2& complex_src2, iu::VolumeGpu_32f_C2& complex_dst){iuprivate::math::complex::multiplyConjugate(complex_src1,complex_src2,complex_dst);}
void multiplyConjugate(iu::ImageCpu_32f_C2& complex_src1, iu::ImageCpu_32f_C2& complex_src2, iu::ImageCpu_32f_C2& complex_dst){iuprivate::math::complex::multiplyConjugate(complex_src1,complex_src2,complex_dst);}
void multiplyConjugate(iu::ImageGpu_32f_C2& complex_src1, iu::ImageGpu_32f_C2& complex_src2, iu::ImageGpu_32f_C2& complex_dst){iuprivate::math::complex::multiplyConjugate(complex_src1,complex_src2,complex_dst);}
void multiplyConjugate(iu::LinearHostMemory_32f_C2& complex_src1, iu::LinearHostMemory_32f_C2& complex_src2, iu::LinearHostMemory_32f_C2& complex_dst){iuprivate::math::complex::multiplyConjugate(complex_src1,complex_src2,complex_dst);}
void multiplyConjugate(iu::LinearDeviceMemory_32f_C2& complex_src1, iu::LinearDeviceMemory_32f_C2& complex_src2, iu::LinearDeviceMemory_32f_C2& complex_dst){iuprivate::math::complex::multiplyConjugate(complex_src1,complex_src2,complex_dst);}

void multiplyConjugate(iu::VolumeCpu_64f_C2& complex_src1, iu::VolumeCpu_64f_C2& complex_src2, iu::VolumeCpu_64f_C2& complex_dst){iuprivate::math::complex::multiplyConjugate(complex_src1,complex_src2,complex_dst);}
void multiplyConjugate(iu::VolumeGpu_64f_C2& complex_src1, iu::VolumeGpu_64f_C2& complex_src2, iu::VolumeGpu_64f_C2& complex_dst){iuprivate::math::complex::multiplyConjugate(complex_src1,complex_src2,complex_dst);}
void multiplyConjugate(iu::ImageCpu_64f_C2& complex_src1, iu::ImageCpu_64f_C2& complex_src2, iu::ImageCpu_64f_C2& complex_dst){iuprivate::math::complex::multiplyConjugate(complex_src1,complex_src2,complex_dst);}
void multiplyConjugate(iu::ImageGpu_64f_C2& complex_src1, iu::ImageGpu_64f_C2& complex_src2, iu::ImageGpu_64f_C2& complex_dst){iuprivate::math::complex::multiplyConjugate(complex_src1,complex_src2,complex_dst);}
void multiplyConjugate(iu::LinearHostMemory_64f_C2& complex_src1, iu::LinearHostMemory_64f_C2& complex_src2, iu::LinearHostMemory_64f_C2& complex_dst){iuprivate::math::complex::multiplyConjugate(complex_src1,complex_src2,complex_dst);}
void multiplyConjugate(iu::LinearDeviceMemory_64f_C2& complex_src1, iu::LinearDeviceMemory_64f_C2& complex_src2, iu::LinearDeviceMemory_64f_C2& complex_dst){iuprivate::math::complex::multiplyConjugate(complex_src1,complex_src2,complex_dst);}

}


} //namespace math
} //namespace iu

