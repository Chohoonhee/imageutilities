#include "hip/hip_runtime.h"

#include "iumath.h"
#include "iucore.h"
#include "iumath/arithmetics.cuh"

namespace iu {
namespace math {

// add constant
void addC(iu::ImageGpu_32f_C1& src, const float& val, iu::ImageGpu_32f_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_32f_C2& src, const float2& val, iu::ImageGpu_32f_C2& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_32f_C3& src, const float3& val, iu::ImageGpu_32f_C3& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_32f_C4& src, const float4& val, iu::ImageGpu_32f_C4& dst) {iuprivate::math::addC(src,val,dst);}

void addC(iu::ImageGpu_32s_C1& src, const int& val, iu::ImageGpu_32s_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_32u_C1& src, const unsigned int& val, iu::ImageGpu_32u_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_16u_C1& src, const unsigned short& val, iu::ImageGpu_16u_C1& dst) {iuprivate::math::addC(src,val,dst);}
void addC(iu::ImageGpu_8u_C1& src, const unsigned char& val, iu::ImageGpu_8u_C1& dst) {iuprivate::math::addC(src,val,dst);}

// multiply constant
void mulC(iu::ImageGpu_32f_C1& src, const float& val, iu::ImageGpu_32f_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_32f_C2& src, const float2& val, iu::ImageGpu_32f_C2& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_32f_C3& src, const float3& val, iu::ImageGpu_32f_C3& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_32f_C4& src, const float4& val, iu::ImageGpu_32f_C4& dst) {iuprivate::math::mulC(src,val,dst);}

void mulC(iu::ImageGpu_32s_C1& src, const int& val, iu::ImageGpu_32s_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_32u_C1& src, const unsigned int& val, iu::ImageGpu_32u_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_16u_C1& src, const unsigned short& val, iu::ImageGpu_16u_C1& dst) {iuprivate::math::mulC(src,val,dst);}
void mulC(iu::ImageGpu_8u_C1& src, const unsigned char& val, iu::ImageGpu_8u_C1& dst) {iuprivate::math::mulC(src,val,dst);}

// pointwise weighted add

void addWeighted(iu::ImageGpu_32f_C1& src1, const float& weight1,
                 iu::ImageGpu_32f_C1& src2, const float& weight2,iu::ImageGpu_32f_C1& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageGpu_32f_C2& src1, const float2& weight1,
                 iu::ImageGpu_32f_C2& src2, const float2& weight2,iu::ImageGpu_32f_C2& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageGpu_32f_C3& src1, const float3& weight1,
                 iu::ImageGpu_32f_C3& src2, const float3& weight2,iu::ImageGpu_32f_C3& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}
void addWeighted(iu::ImageGpu_32f_C4& src1, const float4& weight1,
                 iu::ImageGpu_32f_C4& src2, const float4& weight2,iu::ImageGpu_32f_C4& dst) {iuprivate::math::addWeighted(src1,weight1,src2,weight2,dst);}

} //namespace math
} //namespace iu
