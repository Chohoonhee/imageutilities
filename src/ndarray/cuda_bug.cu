
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

__global__ void my_kernel_00(void * p){
}

int main(){

	int * ptr;
	hipMallocManaged(&ptr, 1000*sizeof(int), hipMemAttachGlobal);

	my_kernel_00 <<< 1, 1 >>>(0);
	//cudaDeviceSynchronize();

	std::cout << ptr[81] << "\n";

	return 0;
}
