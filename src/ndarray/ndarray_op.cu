#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

#include "ndarray_op.cuh"

#include "error_cuda.h"

//! copy
void copy_raw_data(void * dest, void * src, size_t n_bytes){
	hipMemcpy(dest, src, n_bytes, hipMemcpyDefault);
	cuda_check_error();
};

namespace device_op{

//template<typename type, int dims, typename Func>
//void for_each_device(const ndarray_ref<type, dims> & r, Func func){
//	// launch
//	error_text("Kernel launch not implemented");
//};

// a+=val
template<typename type, int dims>
ndarray_ref<type, dims> & operator += (ndarray_ref<type, dims> & a, const type val){
	auto func = [=] __device__ (const intn<dims> & ii){ a.kernel()(ii) += val; }; // operation capture
	struct_dims<dims>::for_each(a.shape(), func);
	return a;
}

// a*=val
template<typename type, int dims>
ndarray_ref<type, dims> & operator *= (ndarray_ref<type, dims> & a, const type val){
	auto func = [=] __device__ (const intn<dims> & ii){ a.kernel()(ii) *= val; }; // operation capture
	struct_dims<dims>::for_each(a.shape(), func);
	return a;
}

// a+=b
template<typename type, int dims>
	ndarray_ref<type, dims> & operator += (ndarray_ref<type, dims> & a, const ndarray_ref<type, dims> & b){
	auto func = [=] __device__ (const intn<dims> & ii){ a.kernel()(ii) += b.kernel()(ii); }; // operation capture
	struct_dims<dims>::for_each(a.shape(), func);
	return a;
}

// a-=b
template<typename type, int dims>
	ndarray_ref<type, dims> & operator -= (ndarray_ref<type, dims> & a, const ndarray_ref<type, dims> & b){
	auto func = [=] __device__ (const intn<dims> & ii){ a.kernel()(ii) -= b.kernel()(ii); }; // operation capture
	struct_dims<dims>::for_each(a.shape(), func);
	return a;
}

// a*=b
template<typename type, int dims>
	ndarray_ref<type, dims> & operator *= (ndarray_ref<type, dims> & a, const ndarray_ref<type, dims> & b){
	auto func = [=] __device__ (const intn<dims> & ii){ a.kernel()(ii) *= b.kernel()(ii); }; // operation capture
	struct_dims<dims>::for_each(a.shape(), func);
	return a;
}

template<typename type, int dims>
ndarray_ref<type, dims> & operator << (ndarray_ref<type, dims> & a, const type val){
	//ckeck_allowed(a);
	auto func = [=] __device__ (const intn<dims> & ii){ a.kernel()(ii) = val; }; // operation capture
	struct_dims<dims>::for_each(a.shape(), func);
	return a;
}

template<typename type, int dims>
ndarray_ref<type, dims> & copy_data (ndarray_ref<type, dims> & a, const ndarray_ref<type, dims> & b){
	//ckeck_allowed(a);
	auto func = [=] __device__ (const intn<dims> & ii){ a.kernel()(ii) = b.kernel()(ii); }; // operation capture
	struct_dims<dims>::for_each(a.shape(), func);
	return a;
};

//! converting operator <<
template<typename type1, typename type2, int dims>
ndarray_ref<type1, dims> & operator << (ndarray_ref<type1, dims> & a, const ndarray_ref<type2, dims> & b){
	auto func = [=] __device__ (const intn<dims> & ii){ a.kernel()(ii) = type1(b.kernel()(ii)); }; // operation capture
	struct_dims<dims>::for_each(a.shape(), func);
	return a;
};

template<typename type, int dims>
ndarray_ref<type, dims> & madd2(ndarray_ref<type, dims> & a, const ndarray_ref<type, dims> & b, const ndarray_ref<type, dims> & c, type w1, type w2){
	auto func = [=] __device__ (const intn<dims> & ii){ a.kernel()(ii) = b.kernel()(ii)*w1 + c.kernel()(ii)*w2; }; // operation capture
	struct_dims<dims>::for_each(a.shape(), func);
	return a;
}


/*
template<typename type, int dims>
const ndarray_ref<type, dims> & operator *= (const ndarray_ref<type, dims> & a, const type val){
	auto func = [=] (const intn<dims> & ii){ a(ii) *= val; };
	auto func_device = [=] __device__ (const intn<dims> & ii){ a(ii) *= val; };
	//auto func = [=] __device__ (const intn<dims> & ii){ a(ii) *= val; };
	//for_each(a, func);
	struct_dims<dims>::for_each_on_device(a.shape(), func_device);
	return a;
}
*/
//template const ndarray_ref<float, 2> & operator *= (const ndarray_ref<float, 2> & a, const float val);
/*
template ndarray_ref<float, 2> & operator *= (ndarray_ref<float, 2> & a, const float val);
template ndarray_ref<float, 2> & operator << (ndarray_ref<float, 2> & a, const float val);
template ndarray_ref<float, 2> & copy_data (ndarray_ref<float, 2> & a, const ndarray_ref<float, 2> & b);

template ndarray_ref<float, 3> & operator *= (ndarray_ref<float, 3> & a, const float val);
template ndarray_ref<float, 3> & operator << (ndarray_ref<float, 3> & a, const float val);
template ndarray_ref<float, 3> & copy_data (ndarray_ref<float, 3> & a, const ndarray_ref<float, 3> & b);
*/

//template ndarray_ref<float,3> & operator += (ndarray_ref<float,3> & a, const ndarray_ref<float,3> & b);

};

template<typename type, int dims>
void ttest(){
	ndarray_ref<type, dims> a, b;
	a *= type(1);
	a += type(1);
	a << type(1);
	a += a;
	a -= a;
	a *= a;
	copy_data(a,b);
	madd2(a,a,a,type(1),type(1));
}

template<typename type>
void tdtest(){
	ttest<type,1>();
	ttest<type,2>();
	ttest<type,3>();
	ttest<type,4>();
}

template<int dims>
void dtest(){
	// conversions
	ndarray_ref<float, dims> f;
	ndarray_ref<int, dims> i;
	ndarray_ref<double, dims> d;
	ndarray_ref<unsigned int, dims> ui;
	ndarray_ref<short int, dims> si;
	f << f;
	i << i;
	i << f;
	f << i;
	d << d;
	f << d;
	d << f;
	ui << ui;
	si << si;
}

void test_ops(){
	tdtest<float>();
	tdtest<int>();
	// conversions
	dtest<1>();
	dtest<2>();
	dtest<3>();
	dtest<4>();
}
