#include "hip/hip_runtime.h"
#ifdef WIN32
// necessary for Intellisense
#include <hip/hip_runtime.h>
#include <>
#endif

// Device code
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < N)
		C[i] = A[i] + B[i];
}

// Interface functions

namespace cuda
{

void launchTestKernel(float *h_A, float *h_B, float *h_C, int N)
{
	int size = N*sizeof(float);

	// Allocate vectors in device memory
	float* d_A;
	hipMalloc(&d_A, size);
	float* d_B;
	hipMalloc(&d_B, size);
	float* d_C;
	hipMalloc(&d_C, size);

	// Copy vectors from host memory to device memory
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	// Invoke kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	VecAdd << <blocksPerGrid, threadsPerBlock >> >(d_A, d_B, d_C, N);

	// Copy result from device memory to host memory
	// h_C contains the result in host memory
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	// Free device memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

}